#include <limits.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <unistd.h>
#include<hip/hip_runtime.h>

#include<time.h>
#include<sys/stat.h>
#include<ctype.h>

__constant__ int d_NumL[2];
__constant__ char d_Pchar[1084];
__constant__ double parameter[5916];
__constant__ float d_stab[4096];
__constant__ float d_deltah[16];
__constant__ float d_deltas[16];

char str2int_CPU(char c)
{
        switch (c)
        {
                case 'A':
                        return 0;
                case 'C':
                        return 1;
                case 'G':
                        return 2;              
                case 'T':  
                        return 3;       
        }
        return 4;
}

__device__ void str2int(char c,char *d_numSeq,int id)
{
        switch (c)
        {
                case 'A':
                        d_numSeq[id]=0;
                        break;
                case 'C':
                        d_numSeq[id]=1;
                        break;
                case 'G':
                        d_numSeq[id]=2;
                        break;
                case 'T':
                        d_numSeq[id]=3;
                        break;
                default:
                        d_numSeq[id]=4;
                        break;
        }
}

__device__ void str2int_rev(char c,char *d_numSeq,int id)
{
        switch (c)
        {
                case 'T':
                        d_numSeq[id]=0;
                        break;
                case 'G':
                        d_numSeq[id]=1;
                        break;
                case 'C':
                        d_numSeq[id]=2;
                        break;                 
                case 'A':               
                        d_numSeq[id]=3;
                        break;
                default:
                        d_numSeq[id]=4;
                        break;
        }
}

void readLoop(FILE *file,double *v1,double *v2,double *v3)
{
        char *line,*p,*q;
        
        line=(char *)malloc(200);
        memset(line,'\0',200);
        fgets(line,200,file);

        p = line;
        while (*p==' '||*p=='\t')
                p++;
        while (*p=='0'||*p=='1'||*p=='2'||*p=='3'||*p=='4'||*p=='5'||*p=='6'||*p=='7'||*p=='8'||*p=='9') 
                p++;
        while (*p==' '||*p=='\t') 
                p++;

        q = p;
        while (!(*q==' '||*q=='\t')) 
                q++;
        *q = '\0';
        q++;
        if (!strcmp(p, "inf"))
                *v1 =1.0*INFINITY;
        else 
                sscanf(p, "%lf", v1);
        while (*q==' '||*q=='\t')
                q++;

        p = q;
        while (!(*p==' '||*p=='\t'))
                p++;
        *p = '\0';
        p++;
        if (!strcmp(q, "inf"))
                *v2 =1.0*INFINITY;
        else 
                sscanf(q, "%lf", v2);
        while (*p==' '||*p=='\t')
                p++;

        q = p;
        while (!(*q==' '||*q=='\t') && (*q != '\0'))
                q++;
        *q = '\0';
        if (!strcmp(p, "inf"))
                *v3 =1.0*INFINITY;
        else 
                sscanf(p, "%lf", v3);
}

void getStack(char *path,double *parameter)
{
        int i, j, ii, jj;
        FILE *sFile, *hFile;
        char *line;

        i=strlen(path)+20;
        line=(char *)malloc(i);
        memset(line,'\0',i);
        strcpy(line,path);
        strcat(line,"stack.ds");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        sFile=fopen(line,"r");
        if(sFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }

        memset(line,'\0',i);
        strcpy(line,path);
        strcat(line,"stack.dh");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        hFile=fopen(line,"r");
        if(hFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }
        free(line);

        line=(char *)malloc(20);
        memset(line,'\0',20);
        for (i = 0; i < 5; ++i)
        {
                for (ii = 0; ii < 5; ++ii)
                {
                        for (j = 0; j < 5; ++j)
                        {
                                for (jj = 0; jj < 5; ++jj)
                                {
                                        if (i == 4 || j == 4 || ii == 4 || jj == 4) //N 
                                        {
                                                parameter[i*125+ii*25+j*5+jj] = -1.0;
                                                parameter[625+i*125+ii*25+j*5+jj]=1.0*INFINITY;
                                        }
                                        else 
                                        {
                                                if(fgets(line,20,sFile)==NULL)
                                                {
                                                        printf("Error! When read parameters in getStack function!\n");
                                                        exit(1);
                                                }
                                                if(strncmp(line, "inf", 3)==0)
                                                        parameter[i*125+ii*25+j*5+jj]=1.0*INFINITY;
                                                else
                                                        parameter[i*125+ii*25+j*5+jj] = atof(line);

                                                if(fgets(line,20,hFile)==NULL)
                                                {
                                                        printf("Error! When read parameters in getStack function!\n");
                                                        exit(1);
                                                }
                                                if(strncmp(line, "inf", 3)==0)
                                                        parameter[625+i*125+ii*25+j*5+jj]=1.0*INFINITY;
                                                else
                                                        parameter[625+i*125+ii*25+j*5+jj] = atof(line);

                                                if (fabs(parameter[i*125+ii*25+j*5+jj])>999999999 ||fabs(parameter[625+i*125+ii*25+j*5+jj])>999999999) 
                                                {
                                                        parameter[i*125+ii*25+j*5+jj] = -1.0;
                                                        parameter[625+i*125+ii*25+j*5+jj] =1.0*INFINITY;
                                                }
                                        }
                                }
                        }
                }
        }
        fclose(sFile);
        fclose(hFile);
        free(line);
}

void getStackint2(char *path,double *parameter)
{
        int i, j, ii, jj;
        FILE *sFile, *hFile;
        char *line;

        i=strlen(path)+20;
        line=(char *)malloc(i);
        memset(line,'\0',i);
        strcpy(line,path);
        strcat(line,"stackmm.ds");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        sFile=fopen(line,"r");
        if(sFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }

        memset(line,'\0',i);
        strcpy(line,path);
        strcat(line,"stackmm.dh");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        hFile=fopen(line,"r");
        if(hFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }
        free(line);

        line=(char *)malloc(20);
        memset(line,'\0',20);
        for (i = 0; i < 5; ++i)
        {
                for (ii = 0; ii < 5; ++ii)
                {
                        for (j = 0; j < 5; ++j)
                        {
                                for (jj = 0; jj < 5; ++jj)
                                {
                                        if (i == 4 || j == 4 || ii == 4 || jj == 4)
                                        {
                                                parameter[1250+i*125+ii*25+j*5+jj] = -1.0;
                                                parameter[1875+i*125+ii*25+j*5+jj] =1.0*INFINITY;
                                        } 
                                        else 
                                        {
                                                if(fgets(line,20,sFile)==NULL)
                                                {
                                                        printf("Error! When read parameters in getStackint2 function!\n");
                                                        exit(1);
                                                }
                                                if(strncmp(line, "inf", 3)==0)
                                                        parameter[1250+i*125+ii*25+j*5+jj]=1.0*INFINITY;
                                                else
                                                        parameter[1250+i*125+ii*25+j*5+jj] = atof(line);

                                                if(fgets(line,20,hFile)==NULL)
                                                {
                                                        printf("Error! When read parameters in getStackint2 function!\n");
                                                        exit(1);
                                                }
                                                if(strncmp(line, "inf", 3)==0)
                                                        parameter[1875+i*125+ii*25+j*5+jj]=1.0*INFINITY;
                                                else
                                                        parameter[1875+i*125+ii*25+j*5+jj] = atof(line);

                                                if(fabs(parameter[1250+i*125+ii*25+j*5+jj])>999999999||fabs(parameter[1875+i*125+ii*25+j*5+jj])>999999999)
                                                {
                                                        parameter[1250+i*125+ii*25+j*5+jj] = -1.0;
                                                        parameter[1875+i*125+ii*25+j*5+jj] =1.0*INFINITY;
                                                }
                                        }
                                }
                        }
                }
        }
        fclose(sFile);
        fclose(hFile);
        free(line);
}

void getDangle(char *path,double *parameter)
{
        int i, j, k;
        FILE *sFile, *hFile;
        char *line;
        
        i=strlen(path)+20;
        line=(char *)malloc(i);
        memset(line,'\0',i);
        strcpy(line,path);
        strcat(line,"dangle.ds");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        sFile=fopen(line,"r");
        if(sFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }

        memset(line,'\0',i);
        strcpy(line,path);
        strcat(line,"dangle.dh");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        hFile=fopen(line,"r");
        if(hFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }
        free(line);

        line=(char *)malloc(20);
        memset(line,'\0',20);
        for (i = 0; i < 5; ++i)
                for (j = 0; j < 5; ++j)
                        for (k = 0; k < 5; ++k) 
                        {
                                if (i == 4 || j == 4) 
                                {
                                        parameter[2500+i*25+k*5+j] = -1.0;
                                        parameter[2625+i*25+k*5+j] =1.0*INFINITY;
                                }
                                else if (k == 4)
                                {
                                        parameter[2500+i*25+k*5+j] = -1.0;
                                        parameter[2625+i*25+k*5+j] =1.0*INFINITY;
                                } 
                                else
                                {
                                        if(fgets(line,20,sFile)==NULL)
                                        {
                                                printf("Error! When read parameters in getDangle function!\n");
                                                exit(1);
                                        }
                                        if(strncmp(line, "inf", 3)==0)
                                                parameter[2500+i*25+k*5+j]=1.0*INFINITY;
                                        else
                                                parameter[2500+i*25+k*5+j]=atof(line);

                                        if(fgets(line,20,hFile)==NULL)
                                        {
                                                printf("Error! When read parameters in getDangle function!\n");        
                                                exit(1);        
                                        }
                                        if(strncmp(line, "inf", 3)==0)        
                                                parameter[2625+i*25+k*5+j]=1.0*INFINITY;           
                                        else        
                                                parameter[2625+i*25+k*5+j]=atof(line);

                                        if(fabs(parameter[2500+i*25+k*5+j])>999999999||fabs(parameter[2625+i*25+k*5+j])>999999999) 
                                        {
                                                parameter[2500+i*25+k*5+j] = -1.0;
                                                parameter[2625+i*25+k*5+j] =1.0*INFINITY;
                                        }
                                }
                        }

        for (i = 0; i < 5; ++i)
                for (j = 0; j < 5; ++j)
                        for (k = 0; k < 5; ++k) 
                        {
                                if (i == 4 || j == 4)
                                {
                                        parameter[2750+i*25+j*5+k] = -1.0;
                                        parameter[2875+i*25+j*5+k] =1.0*INFINITY;
                                } 
                                else if (k == 4) 
                                {
                                        parameter[2750+i*25+j*5+k] = -1.0;
                                        parameter[2875+i*25+j*5+k] =1.0*INFINITY;
                                }
                                else
                                {
                                        if(fgets(line,20,sFile)==NULL)
                                        {
                                                printf("Error! When read parameters in getDangle function!\n");
                                                exit(1);
                                        }
                                        if(strncmp(line, "inf", 3)==0)
                                                parameter[2750+i*25+j*5+k]=1.0*INFINITY;
                                        else
                                                parameter[2750+i*25+j*5+k]=atof(line);

                                        if(fgets(line,20,hFile)==NULL)
                                        {
                                                printf("Error! When read parameters in getDangle function!\n");        
                                                exit(1);        
                                        }
                                        if(strncmp(line, "inf", 3)==0)        
                                                parameter[2875+i*25+j*5+k]=1.0*INFINITY;           
                                        else        
                                                parameter[2875+i*25+j*5+k]=atof(line);

                                        if(fabs(parameter[2750+i*25+j*5+k])>999999999||fabs(parameter[2875+i*25+j*5+k])>999999999)
                                        {
                                                parameter[2750+i*25+j*5+k] = -1.0;
                                                parameter[2875+i*25+j*5+k] =1.0*INFINITY;
                                        }
                                }
                        }
        fclose(sFile);
        fclose(hFile);
        free(line);
}

void getLoop(char *path,double *parameter)
{
        int k;
        FILE *sFile, *hFile;
        char *line;

        k=strlen(path)+20;
        line=(char *)malloc(k);
        memset(line,'\0',k);
        strcpy(line,path);
        strcat(line,"loops.ds");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        sFile=fopen(line,"r");
        if(sFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }

        memset(line,'\0',k);
        strcpy(line,path);
        strcat(line,"loops.dh");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        hFile=fopen(line,"r");
        if(hFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }
        free(line);

        for (k = 0; k < 30; ++k)
        {
                readLoop(sFile, &parameter[3030+k], &parameter[3060+k], &parameter[3000+k]);
                readLoop(hFile, &parameter[3120+k], &parameter[3150+k], &parameter[3090+k]);
        }
        fclose(sFile);
        fclose(hFile);
}

void getTstack(char *path,double *parameter)
{
        int i1, j1, i2, j2;
        FILE *sFile, *hFile;
        char *line;

        i1=strlen(path)+20;
        line=(char *)malloc(i1);
        memset(line,'\0',i1);
        strcpy(line,path);
        strcat(line,"tstack_tm_inf.ds");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        sFile=fopen(line,"r");
        if(sFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }

        memset(line,'\0',i1);
        strcpy(line,path);      
        strcat(line,"tstack.dh");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }             
        hFile=fopen(line,"r");
        if(sFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);   
        }
        free(line);

        line=(char *)malloc(20);
        memset(line,'\0',20);
        for (i1 = 0; i1 < 5; ++i1)
                for (i2 = 0; i2 < 5; ++i2)
                        for (j1 = 0; j1 < 5; ++j1)
                                for (j2 = 0; j2 < 5; ++j2)
                                        if (i1 == 4 || j1 == 4)
                                        {
                                                parameter[3805+i1*125+i2*25+j1*5+j2]=1.0*INFINITY;
                                                parameter[3180+i1*125+i2*25+j1*5+j2] = -1.0;
                                        }
                                        else if (i2 == 4 || j2 == 4)
                                        {
                                                parameter[3180+i1*125+i2*25+j1*5+j2] = 0.00000000001;
                                                parameter[3805+i1*125+i2*25+j1*5+j2] = 0.0;
                                        }
                                        else
                                        {
                                                if(fgets(line,20,sFile)==NULL)
                                                {
                                                        printf("Error! When read parameters in getTstack function!\n");
                                                        exit(1);
                                                }
                                                if(strncmp(line, "inf", 3)==0)
                                                        parameter[3180+i1*125+i2*25+j1*5+j2]=1.0*INFINITY;
                                                else
                                                        parameter[3180+i1*125+i2*25+j1*5+j2]=atof(line);

                                                if(fgets(line,20,hFile)==NULL)
                                                {
                                                        printf("Error! When read parameters in getTstack function!\n");
                                                        exit(1);
                                                }
                                                if(strncmp(line, "inf", 3)==0)
                                                        parameter[3805+i1*125+i2*25+j1*5+j2]=1.0*INFINITY;
                                                else
                                                        parameter[3805+i1*125+i2*25+j1*5+j2]=atof(line);

                                                if (fabs(parameter[3180+i1*125+i2*25+j1*5+j2])>999999999||fabs(parameter[3805+i1*125+i2*25+j1*5+j2])>999999999)
                                                {
                                                        parameter[3180+i1*125+i2*25+j1*5+j2] = -1.0;
                                                        parameter[3805+i1*125+i2*25+j1*5+j2] =1.0*INFINITY;
                                                }
                                        }
        fclose(sFile);
        fclose(hFile);
        free(line);
}

void getTstack2(char *path,double *parameter)
{
        int i1, j1, i2, j2;
        FILE *sFile, *hFile;
        char *line;

        i1=strlen(path)+20;
        line=(char *)malloc(i1);
        memset(line,'\0',i1);
        strcpy(line,path);
        strcat(line,"tstack2.ds");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        sFile=fopen(line,"r");
        if(sFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }

        memset(line,'\0',i1);
        strcpy(line,path);      
        strcat(line,"tstack2.dh");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }             
        hFile=fopen(line,"r");
        if(sFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);   
        }
        free(line);

        line=(char *)malloc(20);
        memset(line,'\0',20);
        for (i1 = 0; i1 < 5; ++i1)
                for (i2 = 0; i2 < 5; ++i2)
                        for (j1 = 0; j1 < 5; ++j1)
                                for (j2 = 0; j2 < 5; ++j2)
                                        if (i1 == 4 || j1 == 4)
                                        {
                                                parameter[5055+i1*125+i2*25+j1*5+j2] =1.0*INFINITY;
                                                parameter[4430+i1*125+i2*25+j1*5+j2] = -1.0;
                                        }
                                        else if (i2 == 4 || j2 == 4)
                                        {
                                                parameter[4430+i1*125+i2*25+j1*5+j2] = 0.00000000001;
                                                parameter[5055+i1*125+i2*25+j1*5+j2] = 0.0;
                                        }
                                        else
                                        {
                                                if(fgets(line,20,sFile)==NULL)
                                                {
                                                        printf("Error! When read parameters in getTstack2 function!\n");
                                                        exit(1);
                                                }
                                                if(strncmp(line, "inf", 3)==0)
                                                        parameter[4430+i1*125+i2*25+j1*5+j2]=1.0*INFINITY;
                                                else
                                                        parameter[4430+i1*125+i2*25+j1*5+j2]=atof(line);

                                                if(fgets(line,20,hFile)==NULL)
                                                {
                                                        printf("Error! When read parameters in getTstack2 function!\n");
                                                        exit(1);
                                                }
                                                if(strncmp(line, "inf", 3)==0)
                                                        parameter[5055+i1*125+i2*25+j1*5+j2]=1.0*INFINITY;
                                                else
                                                        parameter[5055+i1*125+i2*25+j1*5+j2]=atof(line);


                                                if (fabs(parameter[4430+i1*125+i2*25+j1*5+j2])>999999999||fabs(parameter[5055+i1*125+i2*25+j1*5+j2])>999999999)
                                                {
                                                        parameter[4430+i1*125+i2*25+j1*5+j2] = -1.0;
                                                        parameter[5055+i1*125+i2*25+j1*5+j2] =1.0*INFINITY;
                                                }
                                        }
        fclose(sFile);
        fclose(hFile);
        free(line);
}

int get_num_line(char *path,int flag)
{
	FILE *fp;
	int i,size;
	char *line;

	i=strlen(path)+20;
        line=(char *)malloc(i);
        memset(line,'\0',i);
        strcpy(line,path);
	if(flag==0)
	        strcat(line,"triloop.ds");
	else
		strcat(line,"tetraloop.ds");

        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        fp=fopen(line,"r");
        if(fp==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }

	size=0;
	while(fgets(line,i,fp)!=NULL)
		size++;
	return size;
}

void getTriloop(char *path,double *parameter,char *Pchar,int NumL[])
{
        FILE *sFile, *hFile;
        int i,turn;
        char *line,seq[10],value[10];
        
        i=strlen(path)+20;
        line=(char *)malloc(i);
        memset(line,'\0',i);
        strcpy(line,path);
        strcat(line,"triloop.ds");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        sFile=fopen(line,"r");
        if(sFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }
	
	turn=0;
        while(fscanf(sFile,"%s\t%s\n",seq,value)!=EOF)
        {
		for (i=0;i<5;i++)
			Pchar[5*turn+i]=str2int_CPU(seq[i]);
		if(value[0]=='i')
			parameter[5730+turn]=1.0*INFINITY;
		else
			parameter[5730+turn]=atof(value);
		turn++;
        }
        fclose(sFile);

	i=strlen(path)+20;
        memset(line,'\0',i);
        strcpy(line,path);
        strcat(line,"triloop.dh");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        hFile=fopen(line,"r");
        if(hFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }
        free(line);

	turn=0;
        while(fscanf(hFile,"%s\t%s\n",seq,value)!=EOF)
        {
		for(i=0;i<5;i++)
			Pchar[5*NumL[0]+turn*5+i]=str2int_CPU(seq[i]);
		if(value[0]=='i')
			parameter[5730+NumL[0]+turn]=1.0*INFINITY;
		else
			parameter[5730+NumL[0]+turn]=atof(value);
		turn++;
        }
        fclose(hFile);
}

void getTetraloop(char *path,double *parameter,char *Pchar,int NumL[])
{
        FILE *sFile, *hFile;
        int i, turn;
        char *line,seq[10],value[10];

        i=strlen(path)+20;
        line=(char *)malloc(i);
        memset(line,'\0',i);
        strcpy(line,path);
        strcat(line,"tetraloop.ds");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        sFile=fopen(line,"r");
        if(sFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }

	turn=0;
        while(fscanf(sFile,"%s\t%s\n",seq,value)!=EOF)
        {
		for(i=0;i<6;i++)
			Pchar[10*NumL[0]+turn*6+i]=str2int_CPU(seq[i]);
		if(value[0]=='i')
			parameter[5730+2*NumL[0]+turn]=1.0*INFINITY;
		else
			parameter[5730+2*NumL[0]+turn]=atof(value);
		turn++;
        }
        fclose(sFile);

        memset(line,'\0',i);
        strcpy(line,path);
        strcat(line,"tetraloop.dh");
        if(access(line,0)==-1)
        {
                printf("Error! Don't have %s file!\n",line);
                exit(1);
        }
        hFile=fopen(line,"r");
        if(hFile==NULL)
        {
                printf("Error! Can't open the %s file!\n",line);
                exit(1);
        }
        free(line);
        
	turn=0;
        while(fscanf(hFile,"%s\t%s\n",seq,value)!=EOF)
        {
		for(i=0;i<6;i++)
			Pchar[10*NumL[0]+6*NumL[1]+6*turn+i]=str2int_CPU(seq[i]);
		if(value[0]=='i')
			parameter[5730+2*NumL[0]+NumL[1]+turn]=1.0*INFINITY;
		else
			parameter[5730+2*NumL[0]+NumL[1]+turn]=atof(value);
		turn++;
        }
        fclose(hFile);
}

void tableStartATS(double atp_value,double parameter[] )
{
        int i, j;

        for (i = 0; i < 5; ++i)
                for (j = 0; j < 5; ++j)
                        parameter[5680+i*5+j] = 0.00000000001;
        parameter[5680+3] = parameter[5680+15] = atp_value;
}

void tableStartATH(double atp_value,double parameter[])
{
        int i, j;

        for (i = 0; i < 5; ++i)
                for (j = 0; j < 5; ++j)
                        parameter[5705+i*5+j] = 0.0;
        parameter[5705+3] = parameter[5705+15] = atp_value;
}

//end read parameter
__device__ void initMatrix2(int length,double *d_DPT,char *d_numSeq,int id,int *d_ps)
{
	for(d_ps[id*119+104]=1;d_ps[id*119+104]<=length;++d_ps[id*119+104])
		for(d_ps[id*119+105]=d_ps[id*119+104];d_ps[id*119+105]<=length;++d_ps[id*119+105])
			if(d_ps[id*119+105]-d_ps[id*119+104]<4 || (d_numSeq[id*54+d_ps[id*119+104]]+d_numSeq[id*54+d_ps[id*119+105]]!=3))
			{
				d_DPT[id*1331+(d_ps[id*119+104]-1)*(length-1)+d_ps[id*119+105]-1]=1.0*INFINITY;
				d_DPT[id*1331+625+(d_ps[id*119+104]-1)*(length-1)+d_ps[id*119+105]-1]=-1.0;
			}
			else
			{
				d_DPT[id*1331+(d_ps[id*119+104]-1)*(length-1)+d_ps[id*119+105]-1]=0.0;
				d_DPT[id*1331+625+(d_ps[id*119+104]-1)*(length-1)+d_ps[id*119+105]-1]=-3224.0;
			}
}

__device__ void Ss(int i,int j,int k,int length,char *d_numSeq,int id,double *d_DPT)
{
	if(k==2)
	{
		if(i>=j)
		{
			d_DPT[id*1331+1330]=-1.0;
			return;
		}
		if(i==length||j==length+1)
		{
			d_DPT[id*1331+1330]=-1.0;
			return;
		}

		if(i>length)
			i-=length;
		if(j>length)
			j-=length;
		d_DPT[id*1331+1330]=parameter[d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j-1]];
	}
	else
		d_DPT[id*1331+1330]=parameter[d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j+1]];
}

__device__ void Hs(int i,int j,int k,int length,char *d_numSeq,int id,double *d_DPT)
{
	if(k==2)
	{
		if(i>= j)
		{
			d_DPT[id*1331+1330]=1.0*INFINITY;
			return;
		}
		if(i==length||j==length+1)
		{
			d_DPT[id*1331+1330]=1.0*INFINITY;
			return;
		}

		if(i>length)
			i-=length;
		if(j>length)
			j-=length;
		if(fabs(parameter[625+d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j-1]])<999999999)
			d_DPT[id*1331+1330]=parameter[625+d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j-1]];
		else
			d_DPT[id*1331+1330]=1.0*INFINITY;
	}
	else
		d_DPT[id*1331+1330]=parameter[625+d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j+1]];
}

__device__ void maxTM2(int i,int j,int length,double *d_DPT,char *d_numSeq,int id)
{
	d_DPT[id*1331+1314]=d_DPT[id*1331+625+(i-1)*(length-1)+j-1];
	d_DPT[id*1331+1316]=d_DPT[id*1331+(i-1)*(length-1)+j-1];
	d_DPT[id*1331+1312]=(d_DPT[id*1331+1316]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1314]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
	if(fabs(d_DPT[id*1331+(i-1)*(length-1)+j-1])<999999999)
	{
		Ss(i,j,2,length,d_numSeq,id,d_DPT);
		d_DPT[id*1331+1315]=(d_DPT[id*1331+625+i*(length-1)+j-2]+d_DPT[id*1331+1330]);
		Hs(i,j,2,length,d_numSeq,id,d_DPT);
		d_DPT[id*1331+1317]=(d_DPT[id*1331+i*(length-1)+j-2]+d_DPT[id*1331+1330]);
	}
	else
	{
		d_DPT[id*1331+1315]=-1.0;
		d_DPT[id*1331+1317]=1.0*INFINITY;
	}
	d_DPT[id*1331+1313]=(d_DPT[id*1331+1317]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1315]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
	if(d_DPT[id*1331+1315]<-2500.0)
	{
		d_DPT[id*1331+1315]=-3224.0;
		d_DPT[id*1331+1317]=0.0;
	}
	if(d_DPT[id*1331+1314]<-2500.0)
	{
		d_DPT[id*1331+1314]=-3224.0;
		d_DPT[id*1331+1316]=0.0;
 	}

	if(d_DPT[id*1331+1313]>d_DPT[id*1331+1312])
	{
		d_DPT[id*1331+625+(i-1)*(length-1)+j-1]=d_DPT[id*1331+1315];
		d_DPT[id*1331+(i-1)*(length-1)+j-1]= d_DPT[id*1331+1317];
	}
	else
	{
		d_DPT[id*1331+625+(i-1)*(length-1)+j-1]=d_DPT[id*1331+1314];
		d_DPT[id*1331+(i-1)*(length-1)+j-1]=d_DPT[id*1331+1316];
	}
}

__device__ void calc_bulge_internal2(int i,int j,int ii,int jj,int pos,int traceback,int length,double *d_DPT,char *d_numSeq,int id)
{

	d_DPT[id*1331+1318]=-3224.0;
	d_DPT[id*1331+1319]=0.0;

	if(ii-i-2+j-jj>30)
	{
		d_DPT[id*1331+pos]=-1.0;
		d_DPT[id*1331+pos+1]=1.0*INFINITY;
		return;
	}

	if((ii-i-1==0&&j-jj-1>0)||(j-jj-1==0&&ii-i-1>0))
	{
		if(j-jj-1==1||ii-i-1==1)
		{ 
			if((j-jj-1==1&&ii-i-1==0)||(j-jj-1==0&&ii-i-1==1))
			{
				d_DPT[id*1331+1319]=parameter[3150+ii-i+j-jj-3]+parameter[625+d_numSeq[id*54+i]*125+d_numSeq[id*54+ii]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+jj]];
				d_DPT[id*1331+1318]=parameter[3060+ii-i+j-jj-3]+parameter[d_numSeq[id*54+i]*125+d_numSeq[id*54+ii]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+jj]];
 			}
			if(traceback!=1)
			{
				d_DPT[id*1331+1319]+=d_DPT[id*1331+(ii-1)*(length-1)+jj-1];
				d_DPT[id*1331+1318]+=d_DPT[id*1331+625+(ii-1)*(length-1)+jj-1];
			}

			if(fabs(d_DPT[id*1331+1319])>999999999)
			{
				d_DPT[id*1331+1319]=1.0*INFINITY;
				d_DPT[id*1331+1318]=-1.0;
			}
			d_DPT[id*1331+1316]=(d_DPT[id*1331+1319]+d_DPT[id*1331+1302])/((d_DPT[id*1331+1318]+d_DPT[id*1331+1303])+d_DPT[id*1331+1304]);
			d_DPT[id*1331+1317]=(d_DPT[id*1331+(i-1)*(length-1)+j-1]+d_DPT[id*1331+1302])/((d_DPT[id*1331+625+(i-1)*(length-1)+j-1])+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
			if((d_DPT[id*1331+1316]>d_DPT[id*1331+1317])||((traceback&&d_DPT[id*1331+1316]>=d_DPT[id*1331+1317])||traceback==1))
			{
				d_DPT[id*1331+pos]=d_DPT[id*1331+1318];
				d_DPT[id*1331+pos+1]=d_DPT[id*1331+1319];
			}
		}
		else
		{
			d_DPT[id*1331+1319]=parameter[3150+ii-i+j-jj-3]+parameter[5705+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[5705+d_numSeq[id*54+ii]*5+d_numSeq[id*54+27+jj]];
			if(traceback!=1)
				d_DPT[id*1331+1319]+=d_DPT[id*1331+(ii-1)*(length-1)+jj-1];

			d_DPT[id*1331+1318]=parameter[3060+ii-i+j-jj-3]+parameter[5680+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[5680+d_numSeq[id*54+ii]*5+d_numSeq[id*54+27+jj]];
			if(traceback!=1)
				d_DPT[id*1331+1318]+=d_DPT[id*1331+625+(ii-1)*(length-1)+jj-1];
			if(fabs(d_DPT[id*1331+1319])>999999999)
			{
				d_DPT[id*1331+1319]=1.0*INFINITY;
				d_DPT[id*1331+1318]=-1.0;
			}
			d_DPT[id*1331+1316]=(d_DPT[id*1331+1319]+d_DPT[id*1331+1302])/((d_DPT[id*1331+1318]+d_DPT[id*1331+1303])+d_DPT[id*1331+1304]);
			d_DPT[id*1331+1317]=(d_DPT[id*1331+(i-1)*(length-1)+j-1]+d_DPT[id*1331+1302])/(d_DPT[id*1331+625+(i-1)*(length-1)+j-1]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
			if((d_DPT[id*1331+1316]>d_DPT[id*1331+1317])||((traceback&&d_DPT[id*1331+1316]>=d_DPT[id*1331+1317])||(traceback==1)))
			{
				d_DPT[id*1331+pos]=d_DPT[id*1331+1318];
				d_DPT[id*1331+pos+1]=d_DPT[id*1331+1319];
			}
		}
	}
	else if(ii-i-1==1&&j-jj-1==1)
	{
		d_DPT[id*1331+1318]=parameter[1250+d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j-1]]+parameter[1250+d_numSeq[id*54+27+jj]*125+d_numSeq[id*54+27+jj+1]*25+d_numSeq[id*54+ii]*5+d_numSeq[id*54+ii-1]];
		if(traceback!=1)
			d_DPT[id*1331+1318]+=d_DPT[id*1331+625+(ii-1)*(length-1)+jj-1];

		d_DPT[id*1331+1319]=parameter[1875+d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j-1]]+parameter[1875+d_numSeq[id*54+27+jj]*125+d_numSeq[id*54+27+jj+1]*25+d_numSeq[id*54+ii]*5+d_numSeq[id*54+ii-1]];
		if(traceback!=1)
			d_DPT[id*1331+1319]+=d_DPT[id*1331+(ii-1)*(length-1)+jj-1];
		if(fabs(d_DPT[id*1331+1319])>999999999)
		{
			d_DPT[id*1331+1319]=1.0*INFINITY;
			d_DPT[id*1331+1318]=-1.0;
		}
		d_DPT[id*1331+1316]=(d_DPT[id*1331+1319]+d_DPT[id*1331+1302])/((d_DPT[id*1331+1318]+d_DPT[id*1331+1303])+d_DPT[id*1331+1304]);
		d_DPT[id*1331+1317]=(d_DPT[id*1331+(i-1)*(length-1)+j-1]+d_DPT[id*1331+1302])/(d_DPT[id*1331+625+(i-1)*(length-1)+j-1]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
		if((d_DPT[id*1331+1316]-d_DPT[id*1331+1317]>=0.000001)||traceback)
		{
			if((d_DPT[id*1331+1316]>d_DPT[id*1331+1317])||((traceback&&d_DPT[id*1331+1316]>= d_DPT[id*1331+1317])||traceback==1))
			{
				d_DPT[id*1331+pos]=d_DPT[id*1331+1318];
				d_DPT[id*1331+pos+1]=d_DPT[id*1331+1319];
			}
		}
		return;
	}
	else
	{
		d_DPT[id*1331+1319]=parameter[3120+ii-i+j-jj-3]+parameter[3805+d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j-1]]+parameter[3805+d_numSeq[id*54+27+jj]*125+d_numSeq[id*54+27+jj+1]*25+d_numSeq[id*54+ii]*5+d_numSeq[id*54+ii-1]];
		if(traceback!=1)
			d_DPT[id*1331+1319]+=d_DPT[id*1331+(ii-1)*(length-1)+jj-1];

		d_DPT[id*1331+1318]=parameter[3030+ii-i+j-jj-3]+parameter[3180+d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j-1]]+parameter[3180+d_numSeq[id*54+27+jj]*125+d_numSeq[id*54+27+jj+1]*25+d_numSeq[id*54+ii]*5+d_numSeq[id*54+ii-1]]+(-300/310.15*abs(ii-i-j+jj));
		if(traceback!=1)
			d_DPT[id*1331+1318]+=d_DPT[id*1331+625+(ii-1)*(length-1)+jj-1];
		if(fabs(d_DPT[id*1331+1319])>999999999)
		{
			d_DPT[id*1331+1319]=1.0*INFINITY;
			d_DPT[id*1331+1318]=-1.0;
		}

		d_DPT[id*1331+1316]=(d_DPT[id*1331+1319]+d_DPT[id*1331+1302])/((d_DPT[id*1331+1318]+d_DPT[id*1331+1303])+d_DPT[id*1331+1304]);
		d_DPT[id*1331+1317]=(d_DPT[id*1331+(i-1)*(length-1)+j-1]+d_DPT[id*1331+1302])/((d_DPT[id*1331+625+(i-1)*(length-1)+j-1])+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
		if((d_DPT[id*1331+1316]>d_DPT[id*1331+1317])||((traceback&&d_DPT[id*1331+1316]>=d_DPT[id*1331+1317])||(traceback==1)))
		{
			d_DPT[id*1331+pos]=d_DPT[id*1331+1318];
			d_DPT[id*1331+pos+1]=d_DPT[id*1331+1319];
		}
	}
	return;
}

__device__ void CBI(int i,int j,int pos,int traceback,int length,double *d_DPT,char *d_numSeq,int id,int *d_ps)
{
	for(d_ps[id*119+104]=j-i-3;d_ps[id*119+104]>=4&&d_ps[id*119+104]>=j-i-32;--d_ps[id*119+104])
		for(d_ps[id*119+105]=i+1;d_ps[id*119+105]<j-d_ps[id*119+104]&&d_ps[id*119+105]<=length;++d_ps[id*119+105])
		{
			d_ps[id*119+106]=d_ps[id*119+104]+d_ps[id*119+105];
			if(traceback==0)
			{
				d_DPT[id*1331+pos]=-1.0;
				d_DPT[id*1331+pos+1]=1.0*INFINITY;
			}
			if(fabs(d_DPT[id*1331+(d_ps[id*119+105]-1)*(length-1)+d_ps[id*119+106]-1])<999999999)
			{
				calc_bulge_internal2(i,j,d_ps[id*119+105],d_ps[id*119+106],pos,traceback,length,d_DPT,d_numSeq,id);
				if(fabs(d_DPT[id*1331+pos+1])<999999999)
				{
					if(d_DPT[id*1331+pos] <-2500.0)
					{
						d_DPT[id*1331+pos+1]=-3224.0;
						d_DPT[id*1331+pos+1]=0.0;
					}
					if(traceback==0)
					{
						d_DPT[id*1331+(i-1)*(length-1)+j-1]=d_DPT[id*1331+pos+1];
						d_DPT[id*1331+625+(i-1)*(length-1)+j-1]=d_DPT[id*1331+pos];
					}
				}
			}
		}
	return;
}

__device__ void find_pos(char *ref,int ref_start,int start,int length,int num,int *d_ps,int id)
{
	for(d_ps[id*119+105]=0;d_ps[id*119+105]<num;d_ps[id*119+105]++)
	{
		d_ps[id*119+104]=0;
		for(d_ps[id*119+106]=0;d_ps[id*119+106]<length;d_ps[id*119+106]++)
		{
			if(ref[ref_start+d_ps[id*119+106]]!=d_Pchar[start+d_ps[id*119+105]*length+d_ps[id*119+106]])
			{
				d_ps[id*119+104]++;
				break;
			}
		}
		if(d_ps[id*119+104]==0)
		{
			d_ps[id*119+107]=d_ps[id*119+105];
			return;
		}
	}
	d_ps[id*119+107]=-1;
}

__device__ void calc_hairpin(int i,int j,int pos_start,int traceback,int length,double *d_DPT,char *d_numSeq,int id,int *d_ps)
{
	if(j-i<4)
	{
		d_DPT[id*1331+pos_start]=-1.0;
		d_DPT[id*1331+pos_start+1]=1.0*INFINITY;
		return;
	}
	if(i<=length&&length<j)
	{
		d_DPT[id*1331+pos_start]=-1.0;
		d_DPT[id*1331+pos_start+1]=1.0*INFINITY;
		return;
	}
	else if(i>length)
	{
		i-= length;
		j-= length;
	}
	if(j-i-1<=30)
	{
		d_DPT[id*1331+pos_start+1]=parameter[3090+j-i-2];
		d_DPT[id*1331+pos_start]=parameter[3000+j-i-2];
	}
	else
	{
		d_DPT[id*1331+pos_start+1]=parameter[3090+29];
		d_DPT[id*1331+pos_start]=parameter[3000+29];
	}

	if(j-i>4) // for loops 4 bp and more in length, terminal mm are accounted
	{
		d_DPT[id*1331+pos_start+1]+=parameter[5055+d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+j]*5+d_numSeq[id*54+j-1]];
		d_DPT[id*1331+pos_start]+=parameter[4430+d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+j]*5+d_numSeq[id*54+j-1]];
	}
	else if(j-i==4) // for loops 3 bp in length at-penalty is considered
	{
		d_DPT[id*1331+pos_start+1]+=parameter[5705+d_numSeq[id*54+i]*5+d_numSeq[id*54+j]];
		d_DPT[id*1331+pos_start]+=parameter[5680+d_numSeq[id*54+i]*5+d_numSeq[id*54+j]];
	}

	if(j-i-1==3) // closing AT-penalty (+), triloop bonus, hairpin of 3 (+) 
	{
		find_pos(d_numSeq,(id*54+i),5*d_NumL[0],5,d_NumL[0],d_ps,id);
		if(d_ps[id*119+107]!=-1)
			d_DPT[id*1331+pos_start+1]+=parameter[5730+d_NumL[0]+d_ps[id*119+107]];

		find_pos(d_numSeq,(id*54+i),0,5,d_NumL[0],d_ps,id);
		if(d_ps[id*119+107]!=-1)
			d_DPT[id*1331+pos_start]+=parameter[5730+d_ps[id*119+107]];
	}
	else if (j-i-1== 4) // terminal mismatch, tetraloop bonus, hairpin of 4
	{
		find_pos(d_numSeq,(id*54+i),10*d_NumL[0]+6*d_NumL[1],6,d_NumL[1],d_ps,id);
		if(d_ps[id*119+107]!=-1)
			d_DPT[id*1331+pos_start+1]+=parameter[5730+2*d_NumL[0]+d_NumL[1]+d_ps[id*119+107]];

		find_pos(d_numSeq,(id*54+i),10*d_NumL[0],6,d_NumL[1],d_ps,id);
		if(d_ps[id*119+107]!=-1)
			d_DPT[id*1331+pos_start]+=parameter[5730+2*d_NumL[0]+d_ps[id*119+107]];
	}
	if(fabs(d_DPT[id*1331+pos_start+1])>999999999)
	{
		d_DPT[id*1331+pos_start+1] =1.0*INFINITY;
		d_DPT[id*1331+pos_start] = -1.0;
	}
	d_DPT[id*1331+1316]= (d_DPT[id*1331+pos_start+1] +d_DPT[id*1331+1302]) / ((d_DPT[id*1331+pos_start] +d_DPT[id*1331+1303]+ d_DPT[id*1331+1304]));
	d_DPT[id*1331+1317]= (d_DPT[id*1331+(i-1)*(length-1)+j-1] +d_DPT[id*1331+1302]) / ((d_DPT[id*1331+625+(i-1)*(length-1)+j-1]) +d_DPT[id*1331+1303]+ d_DPT[id*1331+1304]);
	if(d_DPT[id*1331+1316]<d_DPT[id*1331+1317] && traceback == 0)
	{
		d_DPT[id*1331+pos_start] =d_DPT[id*1331+625+(i-1)*(length-1)+j-1];
		d_DPT[id*1331+pos_start+1] =d_DPT[id*1331+(i-1)*(length-1)+j-1];
	}
	return;
}

__device__ void fillMatrix2(int length,double *d_DPT,char *d_numSeq,int id,int *d_ps)
{
	for (d_ps[id*119+109] = 2; d_ps[id*119+109] <= length; ++d_ps[id*119+109])
		for (d_ps[id*119+108] = d_ps[id*119+109] - 3 - 1; d_ps[id*119+108] >= 1; --d_ps[id*119+108])
		{
			if (fabs(d_DPT[id*1331+(d_ps[id*119+108]-1)*(length-1)+d_ps[id*119+109]-1])<999999999)
			{
				d_DPT[id*1331+1310] = -1.0;
				d_DPT[id*1331+1311] =1.0*INFINITY;
				maxTM2(d_ps[id*119+108],d_ps[id*119+109],length,d_DPT,d_numSeq,id);
				CBI(d_ps[id*119+108],d_ps[id*119+109],1310,0,length,d_DPT,d_numSeq,id,d_ps);

				d_DPT[id*1331+1310] = -1.0;
				d_DPT[id*1331+1311]=1.0*INFINITY;
				calc_hairpin(d_ps[id*119+108],d_ps[id*119+109],1310,0,length,d_DPT,d_numSeq,id,d_ps);
				if(fabs(d_DPT[id*1331+1311])<999999999)
				{
					if(d_DPT[id*1331+1310]<-2500.0) /* to not give dH any value if dS is unreasonable */
					{
						d_DPT[id*1331+1310]=-3224.0;
						d_DPT[id*1331+1311]= 0.0;
					}
					d_DPT[id*1331+625+(d_ps[id*119+108]-1)*(length-1)+d_ps[id*119+109]-1]=d_DPT[id*1331+1310];
					d_DPT[id*1331+(d_ps[id*119+108]-1)*(length-1)+d_ps[id*119+109]-1]=d_DPT[id*1331+1311];
				}
			}
		}
}

__device__ void max5(double a,double b,double c,double d,double e,int *d_ps,int id)
{
	if(a>b&&a>c&&a>d&&a>e)
		d_ps[id*119+104]=1;
	else if(b>c&&b>d&&b>e)
		d_ps[id*119+104]=2;
	else if(c>d&&c>e)
		d_ps[id*119+104]=3;
	else if(d>e)
		d_ps[id*119+104]=4;
	else
		d_ps[id*119+104]=5;
}

__device__ void END5_1(int i,int hs,int length,double *d_DPT,char *d_numSeq,int id,int *d_ps)
{
	d_DPT[id*1331+1323]=-1.0*INFINITY;
	d_DPT[id*1331+1328]=1.0*INFINITY;
	d_DPT[id*1331+1329]=-1.0;
	for(d_ps[id*119+104]=0;d_ps[id*119+104]<=i-5;++d_ps[id*119+104])
	{
		d_DPT[id*1331+1324]=(d_DPT[id*1331+1276+d_ps[id*119+104]]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1250+d_ps[id*119+104]]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
		d_DPT[id*1331+1325]=d_DPT[id*1331+1302]/(d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
		if(d_DPT[id*1331+1324]>=d_DPT[id*1331+1325])
		{
			d_DPT[id*1331+1326]=d_DPT[id*1331+1276+d_ps[id*119+104]]+parameter[5705+d_numSeq[id*54+d_ps[id*119+104]+1]*5+d_numSeq[id*54+i]]+d_DPT[id*1331+d_ps[id*119+104]*(length-1)+i-1];
			d_DPT[id*1331+1327]=d_DPT[id*1331+1250+d_ps[id*119+104]]+parameter[5680+d_numSeq[id*54+d_ps[id*119+104]+1]*5+d_numSeq[id*54+i]]+d_DPT[id*1331+625+d_ps[id*119+104]*(length-1)+i-1];
			if(fabs(d_DPT[id*1331+1326])>999999999||d_DPT[id*1331+1326]>0||d_DPT[id*1331+1327]>0)  // H and S must be greater than 0 to avoid BS
			{
				d_DPT[id*1331+1326]=1.0*INFINITY;
				d_DPT[id*1331+1327]=-1.0;
			}
			d_DPT[id*1331+1324]=(d_DPT[id*1331+1326]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1327]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
		}
		else
		{
			d_DPT[id*1331+1326]=parameter[5705+d_numSeq[id*54+d_ps[id*119+104]+1]*5+d_numSeq[id*54+i]]+d_DPT[id*1331+d_ps[id*119+104]*(length-1)+i-1];
			d_DPT[id*1331+1327]=parameter[5680+d_numSeq[id*54+d_ps[id*119+104]+1]*5+d_numSeq[id*54+i]]+d_DPT[id*1331+625+d_ps[id*119+104]*(length-1)+i-1];
			if(fabs(d_DPT[id*1331+1326])>999999999||d_DPT[id*1331+1326]>0||d_DPT[id*1331+1327]>0)
			{
				d_DPT[id*1331+1326]=1.0*INFINITY;
				d_DPT[id*1331+1327]=-1.0;
			}
			d_DPT[id*1331+1324]=(d_DPT[id*1331+1326]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1327]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
		}

		if(d_DPT[id*1331+1323]<d_DPT[id*1331+1324])
		{
			if(d_DPT[id*1331+1327]>-2500.0)
			{
				d_DPT[id*1331+1328]=d_DPT[id*1331+1326];
				d_DPT[id*1331+1329]=d_DPT[id*1331+1327];
				d_DPT[id*1331+1323]=d_DPT[id*1331+1324];
			}
		}
	}
	if(hs==1)
		d_DPT[id*1331+1330]=d_DPT[id*1331+1328];
	else
		d_DPT[id*1331+1330]=d_DPT[id*1331+1329];
}

__device__ void END5_2(int i,int hs,int length,double *d_DPT,char *d_numSeq,int id,int *d_ps)
{
	d_DPT[id*1331+1328]=1.0*INFINITY;
	d_DPT[id*1331+1323]=-1.0*INFINITY;
	d_DPT[id*1331+1329]=-1.0;
	for(d_ps[id*119+104]=0;d_ps[id*119+104]<=i-6;++d_ps[id*119+104])
	{
		d_DPT[id*1331+1324]=(d_DPT[id*1331+1276+d_ps[id*119+104]]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1250+d_ps[id*119+104]]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
		d_DPT[id*1331+1325]=d_DPT[id*1331+1302]/(d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
		if(d_DPT[id*1331+1324]>=d_DPT[id*1331+1325])
		{
			d_DPT[id*1331+1326]=d_DPT[id*1331+1276+d_ps[id*119+104]]+parameter[5705+d_numSeq[id*54+d_ps[id*119+104]+2]*5+d_numSeq[id*54+i]]+parameter[2875+d_numSeq[id*54+i]*25+d_numSeq[id*54+d_ps[id*119+104]+2]*5+d_numSeq[id*54+d_ps[id*119+104]+2-1]]+d_DPT[id*1331+(d_ps[id*119+104]+1)*(length-1)+i-1];
			d_DPT[id*1331+1327]=d_DPT[id*1331+1250+d_ps[id*119+104]]+parameter[5680+d_numSeq[id*54+d_ps[id*119+104]+2]*5+d_numSeq[id*54+i]]+parameter[2750+d_numSeq[id*54+i]*25+d_numSeq[id*54+d_ps[id*119+104]+2]*5+d_numSeq[id*54+d_ps[id*119+104]+2-1]]+d_DPT[id*1331+625+(d_ps[id*119+104]+1)*(length-1)+i-1];
			if(fabs(d_DPT[id*1331+1326])>999999999||d_DPT[id*1331+1326]>0||d_DPT[id*1331+1327]>0)
			{
				d_DPT[id*1331+1326]=1.0*INFINITY;
				d_DPT[id*1331+1327]=-1.0;
			}
			d_DPT[id*1331+1324]=(d_DPT[id*1331+1326]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1327]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
		}
		else
		{
			d_DPT[id*1331+1326]=parameter[5705+d_numSeq[id*54+d_ps[id*119+104]+2]*5+d_numSeq[id*54+i]]+parameter[2875+d_numSeq[id*54+i]*25+d_numSeq[id*54+d_ps[id*119+104]+2]*5+d_numSeq[id*54+d_ps[id*119+104]+2-1]]+d_DPT[id*1331+(d_ps[id*119+104]+1)*(length-1)+i-1];
			d_DPT[id*1331+1327]=parameter[5680+d_numSeq[id*54+d_ps[id*119+104]+2]*5+d_numSeq[id*54+i]]+parameter[2750+d_numSeq[id*54+i]*25+d_numSeq[id*54+d_ps[id*119+104]+2]*5+d_numSeq[id*54+d_ps[id*119+104]+2-1]]+d_DPT[id*1331+625+(d_ps[id*119+104]+1)*(length-1)+i-1];
			if(fabs(d_DPT[id*1331+1326])>999999999||d_DPT[id*1331+1326]>0||d_DPT[id*1331+1327]>0)
			{
				d_DPT[id*1331+1326]=1.0*INFINITY;
				d_DPT[id*1331+1327]=-1.0;
			}
			d_DPT[id*1331+1324]=(d_DPT[id*1331+1326]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1327]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
		}

		if(d_DPT[id*1331+1323]<d_DPT[id*1331+1324])
		{
			if(d_DPT[id*1331+1327]>-2500.0)
			{
				d_DPT[id*1331+1328]=d_DPT[id*1331+1326];
				d_DPT[id*1331+1329]=d_DPT[id*1331+1327];
				d_DPT[id*1331+1323]=d_DPT[id*1331+1324];
			}
		}
	}
	if(hs==1)
		d_DPT[id*1331+1330]=d_DPT[id*1331+1328];
	else
		d_DPT[id*1331+1330]=d_DPT[id*1331+1329];
}

__device__ void END5_3(int i,int hs,int length,double *d_DPT,char *d_numSeq,int id,int *d_ps)
{
	d_DPT[id*1331+1328]=1.0*INFINITY;
	d_DPT[id*1331+1323]=-1.0*INFINITY;
	d_DPT[id*1331+1329]=-1.0;
	for(d_ps[id*119+104]=0;d_ps[id*119+104]<=i-6;++d_ps[id*119+104])
	{
		d_DPT[id*1331+1324]=(d_DPT[id*1331+1276+d_ps[id*119+104]]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1250+d_ps[id*119+104]]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
		d_DPT[id*1331+1325]=d_DPT[id*1331+1302]/(d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
		if(d_DPT[id*1331+1324]>=d_DPT[id*1331+1325])
		{
			d_DPT[id*1331+1326]=d_DPT[id*1331+1276+d_ps[id*119+104]]+parameter[5705+d_numSeq[id*54+d_ps[id*119+104]+1]*5+d_numSeq[id*54+i-1]]+parameter[2625+d_numSeq[id*54+i-1]*25+d_numSeq[id*54+i]*5+d_numSeq[id*54+d_ps[id*119+104]+1]]+d_DPT[id*1331+d_ps[id*119+104]*(length-1)+i-2];
			d_DPT[id*1331+1327]=d_DPT[id*1331+1250+d_ps[id*119+104]]+parameter[5680+d_numSeq[id*54+d_ps[id*119+104]+1]*5+d_numSeq[id*54+i-1]]+parameter[2500+d_numSeq[id*54+i-1]*25+d_numSeq[id*54+i]*5+d_numSeq[id*54+d_ps[id*119+104]+1]]+d_DPT[id*1331+625+d_ps[id*119+104]*(length-1)+i-2];
			if(fabs(d_DPT[id*1331+1326])>999999999||d_DPT[id*1331+1326]>0||d_DPT[id*1331+1327]>0)
			{
				d_DPT[id*1331+1326]=1.0*INFINITY;
				d_DPT[id*1331+1327]=-1.0;
			}
			d_DPT[id*1331+1324]=(d_DPT[id*1331+1326]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1327]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
		}
		else
		{
			d_DPT[id*1331+1326]=parameter[5705+d_numSeq[id*54+d_ps[id*119+104]+1]*5+d_numSeq[id*54+i-1]]+parameter[2625+d_numSeq[id*54+i-1]*25+d_numSeq[id*54+i]*5+d_numSeq[id*54+d_ps[id*119+104]+1]]+d_DPT[id*1331+d_ps[id*119+104]*(length-1)+i-2];
			d_DPT[id*1331+1327]=parameter[5680+d_numSeq[id*54+d_ps[id*119+104]+1]*5+d_numSeq[id*54+i-1]]+parameter[2500+d_numSeq[id*54+i-1]*25+d_numSeq[id*54+i]*5+d_numSeq[id*54+d_ps[id*119+104]+1]]+d_DPT[id*1331+625+d_ps[id*119+104]*(length-1)+i-2];
			if(fabs(d_DPT[id*1331+1326])>999999999||d_DPT[id*1331+1326]>0||d_DPT[id*1331+1327]>0)
			{
				d_DPT[id*1331+1326]=1.0*INFINITY;
				d_DPT[id*1331+1327]=-1.0;
			}
			d_DPT[id*1331+1324]=(d_DPT[id*1331+1326]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1327]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
		}

		if(d_DPT[id*1331+1323]<d_DPT[id*1331+1324])
		{
			if(d_DPT[id*1331+1327]>-2500.0)
			{
				d_DPT[id*1331+1328]=d_DPT[id*1331+1326];
				d_DPT[id*1331+1329]=d_DPT[id*1331+1327];
				d_DPT[id*1331+1323]=d_DPT[id*1331+1324];
			}
		}
	}
	if(hs==1)
		d_DPT[id*1331+1330]=d_DPT[id*1331+1328];
	else
		d_DPT[id*1331+1330]=d_DPT[id*1331+1329];
}

__device__ void END5_4(int i,int hs,int length,double *d_DPT,char *d_numSeq,int id,int *d_ps)
{
	d_DPT[id*1331+1328]=1.0*INFINITY;
	d_DPT[id*1331+1323]=-1.0*INFINITY;
	d_DPT[id*1331+1329]=-1.0;
	for(d_ps[id*119+104]=0;d_ps[id*119+104]<=i-7;++d_ps[id*119+104])
	{
		d_DPT[id*1331+1324]=(d_DPT[id*1331+1276+d_ps[id*119+104]]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1250+d_ps[id*119+104]]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
		d_DPT[id*1331+1325]=d_DPT[id*1331+1302]/(d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
		if(d_DPT[id*1331+1324]>=d_DPT[id*1331+1325])
		{
			d_DPT[id*1331+1326]=d_DPT[id*1331+1276+d_ps[id*119+104]]+parameter[5705+d_numSeq[id*54+d_ps[id*119+104]+2]*5+d_numSeq[id*54+i-1]]+parameter[5055+d_numSeq[id*54+i-1]*125+d_numSeq[id*54+i]*25+d_numSeq[id*54+d_ps[id*119+104]+2]*5+d_numSeq[id*54+d_ps[id*119+104]+1]]+d_DPT[id*1331+(d_ps[id*119+104]+1)*(length-1)+i-2];
			d_DPT[id*1331+1327]=d_DPT[id*1331+1250+d_ps[id*119+104]]+parameter[5680+d_numSeq[id*54+d_ps[id*119+104]+2]*5+d_numSeq[id*54+i-1]]+parameter[4430+d_numSeq[id*54+i-1]*125+d_numSeq[id*54+i]*25+d_numSeq[id*54+d_ps[id*119+104]+2]*5+d_numSeq[id*54+d_ps[id*119+104]+1]]+d_DPT[id*1331+625+(d_ps[id*119+104]+1)*(length-1)+i-2];
			if(fabs(d_DPT[id*1331+1326])>999999999||d_DPT[id*1331+1326]>0||d_DPT[id*1331+1327]>0)
			{
				d_DPT[id*1331+1326]=1.0*INFINITY;
				d_DPT[id*1331+1327]=-1.0;
			}
			d_DPT[id*1331+1324]=(d_DPT[id*1331+1326]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1327]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
		}
		else
		{
			d_DPT[id*1331+1326]=parameter[5705+d_numSeq[id*54+d_ps[id*119+104]+2]*5+d_numSeq[id*54+i-1]]+parameter[5055+d_numSeq[id*54+i-1]*125+d_numSeq[id*54+i]*25+d_numSeq[id*54+d_ps[id*119+104]+2]*5+d_numSeq[id*54+d_ps[id*119+104]+1]]+d_DPT[id*1331+(d_ps[id*119+104]+1)*(length-1)+i-2];
			d_DPT[id*1331+1327]=parameter[5680+d_numSeq[id*54+d_ps[id*119+104]+2]*5+d_numSeq[id*54+i-1]]+parameter[4430+d_numSeq[id*54+i-1]*125+d_numSeq[id*54+i]*25+d_numSeq[id*54+d_ps[id*119+104]+2]*5+d_numSeq[id*54+d_ps[id*119+104]+1]]+d_DPT[id*1331+625+(d_ps[id*119+104]+1)*(length-1)+i-2];
			if(fabs(d_DPT[id*1331+1326])>999999999||d_DPT[id*1331+1326]>0||d_DPT[id*1331+1327]>0)
			{
				d_DPT[id*1331+1326]=1.0*INFINITY;
				d_DPT[id*1331+1327]=-1.0;
			}
			d_DPT[id*1331+1324]=(d_DPT[id*1331+1326]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1327]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
 		}

		if(d_DPT[id*1331+1323]<d_DPT[id*1331+1324])
		{
			if(d_DPT[id*1331+1327]>-2500.0)
			{
				d_DPT[id*1331+1328]=d_DPT[id*1331+1326];
				d_DPT[id*1331+1329]=d_DPT[id*1331+1327];
				d_DPT[id*1331+1323]=d_DPT[id*1331+1324];
			}
		}
	}
	if(hs==1)
		d_DPT[id*1331+1330]=d_DPT[id*1331+1328];
	else
		d_DPT[id*1331+1330]=d_DPT[id*1331+1329];
}

__device__ void calc_terminal_bp(double temp,int length,double *d_DPT,char *d_numSeq,int id,int *d_ps)
{
	d_DPT[id*1331+1250+0]=d_DPT[id*1331+1250+1]= -1.0;
	d_DPT[id*1331+1276+0]=d_DPT[id*1331+1276+1]=1.0*INFINITY;

	for(d_ps[id*119+105]=2;d_ps[id*119+105]<=length;d_ps[id*119+105]++)
	{
		d_DPT[id*1331+1250+d_ps[id*119+105]]=-3224.0;
		d_DPT[id*1331+1276+d_ps[id*119+105]]=0;
	}

// adding terminal penalties to 3' end and to 5' end 
	for(d_ps[id*119+105]=2;d_ps[id*119+105]<=length;++d_ps[id*119+105])
	{
		d_DPT[id*1331+1310]=(d_DPT[id*1331+1276+d_ps[id*119+105]-1]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1250+d_ps[id*119+105]-1]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
		END5_1(d_ps[id*119+105],1,length,d_DPT,d_numSeq,id,d_ps);
		d_DPT[id*1331+1315]=d_DPT[id*1331+1330];
		END5_1(d_ps[id*119+105],2,length,d_DPT,d_numSeq,id,d_ps);
		d_DPT[id*1331+1316]=d_DPT[id*1331+1330];
		d_DPT[id*1331+1311]=(d_DPT[id*1331+1315]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1316]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
		END5_2(d_ps[id*119+105],1,length,d_DPT,d_numSeq,id,d_ps);
		d_DPT[id*1331+1317]=d_DPT[id*1331+1330];
		END5_2(d_ps[id*119+105],2,length,d_DPT,d_numSeq,id,d_ps);
		d_DPT[id*1331+1318]=d_DPT[id*1331+1330];
		d_DPT[id*1331+1312]=(d_DPT[id*1331+1317]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1318]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
		END5_3(d_ps[id*119+105],1,length,d_DPT,d_numSeq,id,d_ps);
		d_DPT[id*1331+1319]=d_DPT[id*1331+1330];
		END5_3(d_ps[id*119+105],2,length,d_DPT,d_numSeq,id,d_ps);
		d_DPT[id*1331+1320]=d_DPT[id*1331+1330];
		d_DPT[id*1331+1313]=(d_DPT[id*1331+1319]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1320]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
		END5_4(d_ps[id*119+105],1,length,d_DPT,d_numSeq,id,d_ps);
		d_DPT[id*1331+1321]=d_DPT[id*1331+1330];
		END5_4(d_ps[id*119+105],2,length,d_DPT,d_numSeq,id,d_ps);
		d_DPT[id*1331+1322]=d_DPT[id*1331+1330];
		d_DPT[id*1331+1314]=(d_DPT[id*1331+1321]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1322]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);

		max5(d_DPT[id*1331+1310],d_DPT[id*1331+1311],d_DPT[id*1331+1312],d_DPT[id*1331+1313],d_DPT[id*1331+1314],d_ps,id);
		switch(d_ps[id*119+104])
		{
			case 1:
				d_DPT[id*1331+1250+d_ps[id*119+105]]=d_DPT[id*1331+1250+d_ps[id*119+105]-1];
				d_DPT[id*1331+1276+d_ps[id*119+105]]=d_DPT[id*1331+1276+d_ps[id*119+105]-1];
				break;
			case 2:
				if(d_DPT[id*1331+1315]<temp*d_DPT[id*1331+1316])
				{
					d_DPT[id*1331+1250+d_ps[id*119+105]]=d_DPT[id*1331+1316];
					d_DPT[id*1331+1276+d_ps[id*119+105]]=d_DPT[id*1331+1315];
				}
				else
				{
					d_DPT[id*1331+1250+d_ps[id*119+105]]=d_DPT[id*1331+1250+d_ps[id*119+105]-1];
					d_DPT[id*1331+1276+d_ps[id*119+105]]=d_DPT[id*1331+1276+d_ps[id*119+105]-1];
				}
				break;
			case 3:
				if(d_DPT[id*1331+1317]<temp*d_DPT[id*1331+1318])
				{
					d_DPT[id*1331+1250+d_ps[id*119+105]]=d_DPT[id*1331+1318];
					d_DPT[id*1331+1276+d_ps[id*119+105]]=d_DPT[id*1331+1317];
				}
				else
				{
					d_DPT[id*1331+1250+d_ps[id*119+105]]=d_DPT[id*1331+1250+d_ps[id*119+105]-1];
					d_DPT[id*1331+1276+d_ps[id*119+105]]=d_DPT[id*1331+1276+d_ps[id*119+105]-1];
				}
				break;
			case 4:
				if(d_DPT[id*1331+1319]<temp*d_DPT[id*1331+1320])
				{
					d_DPT[id*1331+1250+d_ps[id*119+105]]=d_DPT[id*1331+1320];
					d_DPT[id*1331+1276+d_ps[id*119+105]]=d_DPT[id*1331+1319];
				}
				else
				{
					d_DPT[id*1331+1250+d_ps[id*119+105]]=d_DPT[id*1331+1250+d_ps[id*119+105]-1];
					d_DPT[id*1331+1276+d_ps[id*119+105]]=d_DPT[id*1331+1276+d_ps[id*119+105]-1];
				}
				break;
			case 5:
				if(d_DPT[id*1331+1321]<temp*d_DPT[id*1331+1322])
				{
					d_DPT[id*1331+1250+d_ps[id*119+105]]=d_DPT[id*1331+1322];
					d_DPT[id*1331+1276+d_ps[id*119+105]]=d_DPT[id*1331+1321];
				}
				else
				{
					d_DPT[id*1331+1250+d_ps[id*119+105]]=d_DPT[id*1331+1250+d_ps[id*119+105]-1];
					d_DPT[id*1331+1276+d_ps[id*119+105]]=d_DPT[id*1331+1276+d_ps[id*119+105]-1];
				}
				break;
			default:
				break;
		}
	}
}

__device__ void newpush(int *d_ps,int id,int i,int j,int mtrx,int total,int next)
{
        for(d_ps[id*119+104]=total-1;d_ps[id*119+104]>=next;d_ps[id*119+104]--)
        {
                d_ps[id*119+50+(d_ps[id*119+104]+1)*3]=d_ps[id*119+50+d_ps[id*119+104]*3];
                d_ps[id*119+50+(d_ps[id*119+104]+1)*3+1]=d_ps[id*119+50+d_ps[id*119+104]*3+1];
                d_ps[id*119+50+(d_ps[id*119+104]+1)*3+2]=d_ps[id*119+50+d_ps[id*119+104]*3+2];
        }
        d_ps[id*119+50+next*3]=i;                  
        d_ps[id*119+50+next*3+1]=j;
        d_ps[id*119+50+next*3+2]=mtrx;
}

__device__ void equal(double a,double b,int *d_ps,int id,int pos)
{
	if(fabs(a)>999999999||fabs(b)>999999999)
	{
		d_ps[id*119+pos]=0;
		return;
	}
	if(fabs(a-b)<1e-5)
		d_ps[id*119+pos]=1;
	else
		d_ps[id*119+pos]=0;
}

__device__ void tracebacku(int *d_ps,int length,double *d_DPT,char *d_numSeq,int id)
{
        newpush(d_ps,id,length,0,1,0,0);
	d_ps[id*119+110]=1;
        d_ps[id*119+111]=0;
        while(d_ps[id*119+111]<d_ps[id*119+110])
        {
                d_ps[id*119+108]=d_ps[id*119+50+3*d_ps[id*119+111]]; // top->i;
                d_ps[id*119+109]=d_ps[id*119+50+3*d_ps[id*119+111]+1]; // top->j;
                if(d_ps[id*119+50+d_ps[id*119+111]*3+2]==1)
                {
			while(1)
			{
				equal(d_DPT[id*1331+1250+d_ps[id*119+108]],d_DPT[id*1331+1250+d_ps[id*119+108]-1],d_ps,id,117);
				equal(d_DPT[id*1331+1276+d_ps[id*119+108]],d_DPT[id*1331+1276+d_ps[id*119+108]-1],d_ps,id,118); // if previous structure is the same as this one
				if(d_ps[id*119+117]&&d_ps[id*119+118])
                                	--d_ps[id*119+108];
				else
					break;
			}
                        if(d_ps[id*119+108]==0)
                                continue;
			END5_1(d_ps[id*119+108],2,length,d_DPT,d_numSeq,id,d_ps);
			d_DPT[id*1331+1329]=d_DPT[id*1331+1330];
			END5_1(d_ps[id*119+108],1,length,d_DPT,d_numSeq,id,d_ps);
                        equal(d_DPT[id*1331+1250+d_ps[id*119+108]],d_DPT[id*1331+1329],d_ps,id,117);
			equal(d_DPT[id*1331+1276+d_ps[id*119+108]],d_DPT[id*1331+1330],d_ps,id,118);
			if(d_ps[id*119+117]&&d_ps[id*119+118])
                        {
                               	for(d_ps[id*119+114]=0;d_ps[id*119+114]<=d_ps[id*119+108]-5;++d_ps[id*119+114])
				{
                               	        equal(d_DPT[id*1331+1250+d_ps[id*119+108]],parameter[5680+d_numSeq[id*54+d_ps[id*119+114]+1]*5+d_numSeq[id*54+d_ps[id*119+108]]]+d_DPT[id*1331+625+d_ps[id*119+114]*(length-1)+d_ps[id*119+108]-1],d_ps,id,117);
					equal(d_DPT[id*1331+1276+d_ps[id*119+108]],parameter[5705+d_numSeq[id*54+d_ps[id*119+114]+1]*5+d_numSeq[id*54+d_ps[id*119+108]]]+d_DPT[id*1331+d_ps[id*119+114]*(length-1)+d_ps[id*119+108]-1],d_ps,id,118);
					if(d_ps[id*119+117]&&d_ps[id*119+118])
                               	        {
                               	                newpush(d_ps,id,d_ps[id*119+114]+1,d_ps[id*119+108],0,d_ps[id*119+110],d_ps[id*119+111]+1);
						d_ps[id*119+110]++;                    
                               	                break;
                               	        }
                               	        else
					{
						equal(d_DPT[id*1331+1250+d_ps[id*119+108]],d_DPT[id*1331+1250+d_ps[id*119+114]]+parameter[5680+d_numSeq[id*54+d_ps[id*119+114]+1]*5+d_numSeq[id*54+d_ps[id*119+108]]]+d_DPT[id*1331+625+d_ps[id*119+114]*(length-1)+d_ps[id*119+108]-1],d_ps,id,117);
						equal(d_DPT[id*1331+1276+d_ps[id*119+108]],d_DPT[id*1331+1276+d_ps[id*119+114]]+parameter[5705+d_numSeq[id*54+d_ps[id*119+114]+1]*5+d_numSeq[id*54+d_ps[id*119+108]]]+d_DPT[id*1331+d_ps[id*119+114]*(length-1)+d_ps[id*119+108]-1],d_ps,id,118);
						if(d_ps[id*119+117]&&d_ps[id*119+118])
	                               	        {
	                               	                newpush(d_ps,id,d_ps[id*119+114]+1,d_ps[id*119+108],0,d_ps[id*119+110],d_ps[id*119+111]+1);
							d_ps[id*119+110]++;
	                               	                newpush(d_ps,id,d_ps[id*119+114],0,1,d_ps[id*119+110],d_ps[id*119+111]+1);
							d_ps[id*119+110]++;
	                               	                break;
	                               	        }
					}
				}
                        }
                        else
			{
				END5_2(d_ps[id*119+108],2,length,d_DPT,d_numSeq,id,d_ps);
				d_DPT[id*1331+1329]=d_DPT[id*1331+1330];
				END5_2(d_ps[id*119+108],1,length,d_DPT,d_numSeq,id,d_ps);
				equal(d_DPT[id*1331+1250+d_ps[id*119+108]],d_DPT[id*1331+1329],d_ps,id,117);
				equal(d_DPT[id*1331+1276+d_ps[id*119+108]],d_DPT[id*1331+1330],d_ps,id,118);
				if(d_ps[id*119+117]&&d_ps[id*119+118])
                        	{
                                	for (d_ps[id*119+114]=0;d_ps[id*119+114]<=d_ps[id*119+108]-6;++d_ps[id*119+114])
					{
                                	        equal(d_DPT[id*1331+1250+d_ps[id*119+108]],parameter[5680+d_numSeq[id*54+d_ps[id*119+114]+2]*5+d_numSeq[id*54+d_ps[id*119+108]]]+parameter[2750+d_numSeq[id*54+d_ps[id*119+108]]*25+d_numSeq[id*54+d_ps[id*119+114]+2]*5+d_numSeq[id*54+d_ps[id*119+114]+2-1]]+d_DPT[id*1331+625+(d_ps[id*119+114]+1)*(length-1)+d_ps[id*119+108]-1],d_ps,id,117);
						equal(d_DPT[id*1331+1276+d_ps[id*119+108]],parameter[5705+d_numSeq[id*54+d_ps[id*119+114]+2]*5+d_numSeq[id*54+d_ps[id*119+108]]]+parameter[2875+d_numSeq[id*54+d_ps[id*119+108]]*25+d_numSeq[id*54+d_ps[id*119+114]+2]*5+d_numSeq[id*54+d_ps[id*119+114]+2-1]]+d_DPT[id*1331+(d_ps[id*119+114]+1)*(length-1)+d_ps[id*119+108]-1],d_ps,id,118);
						if(d_ps[id*119+117]&&d_ps[id*119+118])
                                	        {
                                	                newpush(d_ps,id,d_ps[id*119+114]+2,d_ps[id*119+108],0,d_ps[id*119+110],d_ps[id*119+111]+1);
							d_ps[id*119+110]++;
                                	                break;
                                	        }
                                	        else
						{
							equal(d_DPT[id*1331+1250+d_ps[id*119+108]],d_DPT[id*1331+1250+d_ps[id*119+114]]+parameter[5680+d_numSeq[id*54+d_ps[id*119+114]+2]*5+d_numSeq[id*54+d_ps[id*119+108]]]+parameter[2750+d_numSeq[id*54+d_ps[id*119+108]]*25+d_numSeq[id*54+d_ps[id*119+114]+2]*5+d_numSeq[id*54+d_ps[id*119+114]+2-1]]+d_DPT[id*1331+625+(d_ps[id*119+114]+1)*(length-1)+d_ps[id*119+108]-1],d_ps,id,117);
							equal(d_DPT[id*1331+1276+d_ps[id*119+108]],d_DPT[id*1331+1276+d_ps[id*119+114]]+parameter[5705+d_numSeq[id*54+d_ps[id*119+114]+2]*5+d_numSeq[id*54+d_ps[id*119+108]]]+parameter[2875+d_numSeq[id*54+d_ps[id*119+108]]*25+d_numSeq[id*54+d_ps[id*119+114]+2]*5+d_numSeq[id*54+d_ps[id*119+114]+1]]+d_DPT[id*1331+(d_ps[id*119+114]+1)*(length-1)+d_ps[id*119+108]-1],d_ps,id,118);
							if(d_ps[id*119+117]&&d_ps[id*119+118])
	                                	        {
	                                	                newpush(d_ps,id,d_ps[id*119+114]+2,d_ps[id*119+108],0,d_ps[id*119+110],d_ps[id*119+111]+1);
								d_ps[id*119+110]++;
	                                	                newpush(d_ps,id,d_ps[id*119+114],0,1,d_ps[id*119+110],d_ps[id*119+111]+1);
								d_ps[id*119+110]++;
	                                	                break;
	                                	        }
						}
					}
                        	}
				else
				{
					END5_3(d_ps[id*119+108],2,length,d_DPT,d_numSeq,id,d_ps);
					d_DPT[id*1331+1329]=d_DPT[id*1331+1330];
					END5_3(d_ps[id*119+108],1,length,d_DPT,d_numSeq,id,d_ps);
                        		equal(d_DPT[id*1331+1250+d_ps[id*119+108]],d_DPT[id*1331+1329],d_ps,id,117);
					equal(d_DPT[id*1331+1276+d_ps[id*119+108]],d_DPT[id*1331+1330],d_ps,id,118);
					if(d_ps[id*119+117]&&d_ps[id*119+118])
		                        {
		                                for (d_ps[id*119+114]=0;d_ps[id*119+114]<=d_ps[id*119+108]-6;++d_ps[id*119+114])
						{
		                                        equal(d_DPT[id*1331+1250+d_ps[id*119+108]],parameter[5680+d_numSeq[id*54+d_ps[id*119+114]+1]*5+d_numSeq[id*54+d_ps[id*119+108]-1]]+parameter[2500+d_numSeq[id*54+d_ps[id*119+108]-1]*25+d_numSeq[id*54+d_ps[id*119+108]]*5+d_numSeq[id*54+d_ps[id*119+114]+1]]+d_DPT[id*1331+625+d_ps[id*119+114]*(length-1)+d_ps[id*119+108]-2],d_ps,id,117);
							equal(d_DPT[id*1331+1276+d_ps[id*119+108]],parameter[5705+d_numSeq[id*54+d_ps[id*119+114]+1]*5+d_numSeq[id*54+d_ps[id*119+108]-1]]+parameter[2625+d_numSeq[id*54+d_ps[id*119+108]-1]*25+d_numSeq[id*54+d_ps[id*119+108]]*5+d_numSeq[id*54+d_ps[id*119+114]+1]]+d_DPT[id*1331+d_ps[id*119+114]*(length-1)+d_ps[id*119+108]-2],d_ps,id,118);
							if(d_ps[id*119+117]&&d_ps[id*119+118])
		                                        {
		                                                newpush(d_ps,id,d_ps[id*119+114]+1,d_ps[id*119+108]-1,0,d_ps[id*119+110],d_ps[id*119+111]+1);
								d_ps[id*119+110]++;
		                                                break;
		                                        }
		                                        else
							{
								equal(d_DPT[id*1331+1250+d_ps[id*119+108]],d_DPT[id*1331+1250+d_ps[id*119+114]]+parameter[5680+d_numSeq[id*54+d_ps[id*119+114]+1]*5+d_numSeq[id*54+d_ps[id*119+108]-1]]+parameter[2500+d_numSeq[id*54+d_ps[id*119+108]-1]*25+d_numSeq[id*54+d_ps[id*119+108]]*5+d_numSeq[id*54+d_ps[id*119+114]+1]]+d_DPT[id*1331+625+d_ps[id*119+114]*(length-1)+d_ps[id*119+108]-2],d_ps,id,117);
								equal(d_DPT[id*1331+1276+d_ps[id*119+108]],d_DPT[id*1331+1276+d_ps[id*119+114]]+parameter[5705+d_numSeq[id*54+d_ps[id*119+114]+1]*5+d_numSeq[id*54+d_ps[id*119+108]-1]]+parameter[2625+d_numSeq[id*54+d_ps[id*119+108]-1]*25+d_numSeq[id*54+d_ps[id*119+108]]*5+d_numSeq[id*54+d_ps[id*119+114]+1]]+d_DPT[id*1331+d_ps[id*119+114]*(length-1)+d_ps[id*119+108]-2],d_ps,id,118);
								if(d_ps[id*119+117]&&d_ps[id*119+118])
	        		                                {
	                		                                newpush(d_ps,id,d_ps[id*119+114]+1,d_ps[id*119+108]-1,0,d_ps[id*119+110],d_ps[id*119+111]+1);
									d_ps[id*119+110]++;
	                		                                newpush(d_ps,id,d_ps[id*119+114],0,1,d_ps[id*119+110],d_ps[id*119+111]+1);
									d_ps[id*119+110]++;
	                		                                break;
	                		                        }
							}
						}
                		        }
		                        else
					{
						END5_4(d_ps[id*119+108],2,length,d_DPT,d_numSeq,id,d_ps);
						d_DPT[id*1331+1329]=d_DPT[id*1331+1330];
						END5_4(d_ps[id*119+108],1,length,d_DPT,d_numSeq,id,d_ps);
						equal(d_DPT[id*1331+1250+d_ps[id*119+108]],d_DPT[id*1331+1329],d_ps,id,117);
						equal(d_DPT[id*1331+1276+d_ps[id*119+108]],d_DPT[id*1331+1330],d_ps,id,118);
						if(d_ps[id*119+117]&&d_ps[id*119+118])
			                        {
			                                for (d_ps[id*119+114]=0;d_ps[id*119+114]<=d_ps[id*119+108]-7;++d_ps[id*119+114])
							{
			                                        equal(d_DPT[id*1331+1250+d_ps[id*119+108]],parameter[5680+d_numSeq[id*54+d_ps[id*119+114]+2]*5+d_numSeq[id*54+d_ps[id*119+108]-1]]+parameter[4430+d_numSeq[id*54+d_ps[id*119+108]-1]*125+d_numSeq[id*54+d_ps[id*119+108]]*25+d_numSeq[id*54+d_ps[id*119+114]+2]*5+d_numSeq[id*54+d_ps[id*119+114]+1]]+d_DPT[id*1331+625+(d_ps[id*119+114]+1)*(length-1)+d_ps[id*119+108]-2],d_ps,id,117);
								equal(d_DPT[id*1331+1276+d_ps[id*119+108]],parameter[5705+d_numSeq[id*54+d_ps[id*119+114]+2]*5+d_numSeq[id*54+d_ps[id*119+108]-1]]+parameter[5055+d_numSeq[id*54+d_ps[id*119+108]-1]*125+d_numSeq[id*54+d_ps[id*119+108]]*25+d_numSeq[id*54+d_ps[id*119+114]+2]*5+d_numSeq[id*54+d_ps[id*119+114]+1]]+d_DPT[id*1331+(d_ps[id*119+114]+1)*(length-1)+d_ps[id*119+108]-2],d_ps,id,118);
								if(d_ps[id*119+117]&&d_ps[id*119+118])
			                                        {
			                                                newpush(d_ps,id,d_ps[id*119+114]+2,d_ps[id*119+108]-1,0,d_ps[id*119+110],d_ps[id*119+111]+1);
									d_ps[id*119+110]++;
			                                                break;
			                                        }
			                                        else
								{
									equal(d_DPT[id*1331+1250+d_ps[id*119+108]],d_DPT[id*1331+1250+d_ps[id*119+114]]+parameter[5680+d_numSeq[id*54+d_ps[id*119+114]+2]*5+d_numSeq[id*54+d_ps[id*119+108]-1]]+parameter[4430+d_numSeq[id*54+d_ps[id*119+108]-1]*125+d_numSeq[id*54+d_ps[id*119+108]]*25+d_numSeq[id*54+d_ps[id*119+114]+2]*5+d_numSeq[id*54+d_ps[id*119+114]+1]]+d_DPT[id*1331+625+(d_ps[id*119+114]+1)*(length-1)+d_ps[id*119+108]-2],d_ps,id,117);
									equal(d_DPT[id*1331+1276+d_ps[id*119+108]],d_DPT[id*1331+1276+d_ps[id*119+114]]+parameter[5705+d_numSeq[id*54+d_ps[id*119+114]+2]*5+d_numSeq[id*54+d_ps[id*119+108]-1]]+parameter[5055+d_numSeq[id*54+d_ps[id*119+108]-1]*125+d_numSeq[id*54+d_ps[id*119+108]]*25+d_numSeq[id*54+d_ps[id*119+114]+2]*5+d_numSeq[id*54+d_ps[id*119+114]+1]]+d_DPT[id*1331+(d_ps[id*119+114]+1)*(length-1)+d_ps[id*119+108]-2],d_ps,id,118);
									if(d_ps[id*119+117]&&d_ps[id*119+118])
				                                        {
				                                                newpush(d_ps,id,d_ps[id*119+114]+2,d_ps[id*119+108]-1,0,d_ps[id*119+110],d_ps[id*119+111]+1);
										d_ps[id*119+110]++;
				                                                newpush(d_ps,id,d_ps[id*119+114],0,1,d_ps[id*119+110],d_ps[id*119+111]+1);
										d_ps[id*119+110]++;
				                                                break;
				                                        }
								}
							}
						}
		                        }
				}
			}
                }
                else if(d_ps[id*119+50+3*d_ps[id*119+111]+2]==0)
                {
                        d_ps[id*119+d_ps[id*119+108]-1]=d_ps[id*119+109];
                        d_ps[id*119+d_ps[id*119+109]-1]=d_ps[id*119+108];
                        d_DPT[id*1331+1310]=-1.0;
                        d_DPT[id*1331+1311]=1.0*INFINITY;
                        calc_hairpin(d_ps[id*119+108],d_ps[id*119+109],1310,1,length,d_DPT,d_numSeq,id,d_ps);

                        d_DPT[id*1331+1312]=-1.0;
                        d_DPT[id*1331+1313]=1.0*INFINITY;
                        CBI(d_ps[id*119+108],d_ps[id*119+109],1312,2,length,d_DPT,d_numSeq,id,d_ps);

			Ss(d_ps[id*119+108],d_ps[id*119+109],2,length,d_numSeq,id,d_DPT);
                        equal(d_DPT[id*1331+625+(d_ps[id*119+108]-1)*(length-1)+d_ps[id*119+109]-1],d_DPT[id*1331+1330]+d_DPT[id*1331+625+d_ps[id*119+108]*(length-1)+d_ps[id*119+109]-2],d_ps,id,117);
			if(d_ps[id*119+117])
			{
				Hs(d_ps[id*119+108],d_ps[id*119+109],2,length,d_numSeq,id,d_DPT);
				equal(d_DPT[id*1331+(d_ps[id*119+108]-1)*(length-1)+d_ps[id*119+109]-1],d_DPT[id*1331+1330]+d_DPT[id*1331+d_ps[id*119+108]*(length-1)+d_ps[id*119+109]-2],d_ps,id,118);
				if(d_ps[id*119+118])
                                	newpush(d_ps,id,d_ps[id*119+108]+1,d_ps[id*119+109]-1,0,d_ps[id*119+110],d_ps[id*119+111]+1);
				d_ps[id*119+110]++;
			}
                        else
			{
				equal(d_DPT[id*1331+625+(d_ps[id*119+108]-1)*(length-1)+d_ps[id*119+109]-1],d_DPT[id*1331+1312],d_ps,id,117);
				equal(d_DPT[id*1331+(d_ps[id*119+108]-1)*(length-1)+d_ps[id*119+109]-1],d_DPT[id*1331+1313],d_ps,id,118);
				if(d_ps[id*119+117]&&d_ps[id*119+118])
	                        {
	                                for (d_ps[id*119+116]=0,d_ps[id*119+115]=d_ps[id*119+109]-d_ps[id*119+108]-3;d_ps[id*119+115]>=4&&d_ps[id*119+115]>=d_ps[id*119+109]-d_ps[id*119+108]-32&&!d_ps[id*119+116];--d_ps[id*119+115])
	                                        for (d_ps[id*119+112]=d_ps[id*119+108]+1;d_ps[id*119+112]<d_ps[id*119+109]-d_ps[id*119+115];++d_ps[id*119+112])
	                                        {
	                                                d_ps[id*119+113]=d_ps[id*119+115]+d_ps[id*119+112];
	                                                d_DPT[id*1331+1314]=-1.0;
	                                                d_DPT[id*1331+1315]=1.0*INFINITY;
	                                                calc_bulge_internal2(d_ps[id*119+108],d_ps[id*119+109],d_ps[id*119+112],d_ps[id*119+113],1314,1,length,d_DPT,d_numSeq,id);

	                                                equal(d_DPT[id*1331+625+(d_ps[id*119+108]-1)*(length-1)+d_ps[id*119+109]-1],d_DPT[id*1331+1314]+d_DPT[id*1331+625+(d_ps[id*119+112]-1)*(length-1)+d_ps[id*119+113]-1],d_ps,id,117);
							equal(d_DPT[id*1331+(d_ps[id*119+108]-1)*(length-1)+d_ps[id*119+109]-1],d_DPT[id*1331+1315]+d_DPT[id*1331+(d_ps[id*119+112]-1)*(length-1)+d_ps[id*119+113]-1],d_ps,id,118);
							if(d_ps[id*119+117]&&d_ps[id*119+118])
	                                                {
	                                                        newpush(d_ps,id,d_ps[id*119+112],d_ps[id*119+113],0,d_ps[id*119+110],d_ps[id*119+111]+1);
								d_ps[id*119+110]++;
	                                                        ++d_ps[id*119+116];
	                                                        break;
	                                                }
	                                        }
	                        }
			}
                }
                d_ps[id*119+111]++;
        }
}

__device__ void drawHairpin(int *d_ps,int id,double mh,double ms,int length,double *d_DPT)
{
        d_ps[id*119+105]=0;
        if(fabs(ms)>999999999||fabs(mh)>999999999)
		d_DPT[id*1331+1309]=0.0;
        else
        {
		for(d_ps[id*119+104]=1;d_ps[id*119+104]<length;++d_ps[id*119+104])
		{
			if(d_ps[id*119+d_ps[id*119+104]-1]>0)
				d_ps[id*119+105]++;
                }
                d_DPT[id*1331+1309]=mh/(ms+(((d_ps[id*119+105]/2)-1)*-0.51986))-273.15;
        }
}

__device__ void initMatrix(int length,double *d_DPT,char *d_numSeq,int id,int *d_ps)
{
	for(d_ps[id*119+104]=1;d_ps[id*119+104]<=length;++d_ps[id*119+104])
	{
		for(d_ps[id*119+105]=1;d_ps[id*119+105]<=length;++d_ps[id*119+105])
		{
			if(d_numSeq[id*54+d_ps[id*119+104]]+d_numSeq[id*54+27+d_ps[id*119+105]]!=3)
			{
				d_DPT[id*1331+(d_ps[id*119+104]-1)*length+d_ps[id*119+105]-1]=1.0*INFINITY;
				d_DPT[id*1331+625+(d_ps[id*119+104]-1)*length+d_ps[id*119+105]-1]=-1.0;
			}
			else
			{
				d_DPT[id*1331+(d_ps[id*119+104]-1)*length+d_ps[id*119+105]-1]=0.0;
				d_DPT[id*1331+625+(d_ps[id*119+104]-1)*length+d_ps[id*119+105]-1]=-3224.0;
			}
		}
	}
}

__device__ void LSH(int i,int j,int length,double *d_DPT,char *d_numSeq,int id)
{
	if(d_numSeq[id*54+i]+d_numSeq[id*54+27+j]!=3)
	{
		d_DPT[id*1331+625+(i-1)*length+j-1]=-1.0;
		d_DPT[id*1331+(i-1)*length+j-1]=1.0*INFINITY;
		return;
	}

	d_DPT[id*1331+1312]=parameter[5680+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[4430+d_numSeq[id*54+27+j]*125+d_numSeq[id*54+27+j-1]*25+d_numSeq[id*54+i]*5+d_numSeq[id*54+i-1]];
	d_DPT[id*1331+1313]=parameter[5705+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[5055+d_numSeq[id*54+27+j]*125+d_numSeq[id*54+27+j-1]*25+d_numSeq[id*54+i]*5+d_numSeq[id*54+i-1]];
	if(fabs(d_DPT[id*1331+1313])>999999999)
	{
		d_DPT[id*1331+1313]=1.0*INFINITY;
		d_DPT[id*1331+1312]=-1.0;
	}
// If there is two dangling ends at the same end of duplex
	if(fabs(parameter[2625+d_numSeq[id*54+27+j]*25+d_numSeq[id*54+27+j-1]*5+d_numSeq[id*54+i]])<999999999&&fabs(parameter[2875+d_numSeq[id*54+27+j]*25+d_numSeq[id*54+i]*5+d_numSeq[id*54+i-1]])<999999999)
	{
		d_DPT[id*1331+1315]=parameter[5680+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[2500+d_numSeq[id*54+27+j]*25+d_numSeq[id*54+27+j-1]*5+d_numSeq[id*54+i]]+parameter[2750+d_numSeq[id*54+27+j]*25+d_numSeq[id*54+i]*5+d_numSeq[id*54+i-1]];
		d_DPT[id*1331+1316]=parameter[5705+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[2625+d_numSeq[id*54+27+j]*25+d_numSeq[id*54+27+j-1]*5+d_numSeq[id*54+i]]+parameter[2875+d_numSeq[id*54+27+j]*25+d_numSeq[id*54+i]*5+d_numSeq[id*54+i-1]];
		if(fabs(d_DPT[id*1331+1316])>999999999)
		{
			d_DPT[id*1331+1316]=1.0*INFINITY;
			d_DPT[id*1331+1315]=-1.0;
		}
		d_DPT[id*1331+1317]=(d_DPT[id*1331+1316]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1315]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
		if(fabs(d_DPT[id*1331+1313])<999999999)
		{
			d_DPT[id*1331+1314]=(d_DPT[id*1331+1313]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1312]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
			if(d_DPT[id*1331+1314]<d_DPT[id*1331+1317])
			{
				d_DPT[id*1331+1312]=d_DPT[id*1331+1315];
				d_DPT[id*1331+1313]=d_DPT[id*1331+1316];
				d_DPT[id*1331+1314]=d_DPT[id*1331+1317];
			}
		}
		else
		{
			d_DPT[id*1331+1312]=d_DPT[id*1331+1315];
			d_DPT[id*1331+1313]=d_DPT[id*1331+1316];
			d_DPT[id*1331+1314]=d_DPT[id*1331+1317];
		}
	}
	else if(fabs(parameter[2625+d_numSeq[id*54+27+j]*25+d_numSeq[id*54+27+j-1]*5+d_numSeq[id*54+i]])<999999999)
	{
		d_DPT[id*1331+1315]=parameter[5680+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[2500+d_numSeq[id*54+27+j]*25+d_numSeq[id*54+27+j-1]*5+d_numSeq[id*54+i]];
		d_DPT[id*1331+1316]=parameter[5705+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[2625+d_numSeq[id*54+27+j]*25+d_numSeq[id*54+27+j-1]*5+d_numSeq[id*54+i]];
		if(fabs(d_DPT[id*1331+1316])>999999999)
		{
			d_DPT[id*1331+1316]=1.0*INFINITY;
			d_DPT[id*1331+1315]=-1.0;
		}
		d_DPT[id*1331+1317]=(d_DPT[id*1331+1316]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1315]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
		if(fabs(d_DPT[id*1331+1313])<999999999)
		{
			d_DPT[id*1331+1314]=(d_DPT[id*1331+1313]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1312]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
			if(d_DPT[id*1331+1314]<d_DPT[id*1331+1317])
			{
				d_DPT[id*1331+1312]=d_DPT[id*1331+1315];
				d_DPT[id*1331+1313]=d_DPT[id*1331+1316];
				d_DPT[id*1331+1314]=d_DPT[id*1331+1317];
			}
		}
		else
		{
			d_DPT[id*1331+1312]=d_DPT[id*1331+1315];
			d_DPT[id*1331+1313]=d_DPT[id*1331+1316];
			d_DPT[id*1331+1314]=d_DPT[id*1331+1317];
		}
	}
	else if(fabs(parameter[2875+d_numSeq[id*54+27+j]*25+d_numSeq[id*54+i]*5+d_numSeq[id*54+i-1]])<999999999)
	{
		d_DPT[id*1331+1315]=parameter[5680+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[2750+d_numSeq[id*54+27+j]*25+d_numSeq[id*54+i]*5+d_numSeq[id*54+i-1]];
		d_DPT[id*1331+1316]=parameter[5705+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[2875+d_numSeq[id*54+27+j]*25+d_numSeq[id*54+i]*5+d_numSeq[id*54+i-1]];
		if(fabs(d_DPT[id*1331+1316])>999999999)
		{
			d_DPT[id*1331+1316]=1.0*INFINITY;
			d_DPT[id*1331+1315]=-1.0;
		}
		d_DPT[id*1331+1317]=(d_DPT[id*1331+1316]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1315]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
		if(fabs(d_DPT[id*1331+1313])<999999999)
		{
			d_DPT[id*1331+1314]=(d_DPT[id*1331+1313]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1312]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
			if(d_DPT[id*1331+1314]<d_DPT[id*1331+1317])
			{
				d_DPT[id*1331+1312]=d_DPT[id*1331+1315];
				d_DPT[id*1331+1313]=d_DPT[id*1331+1316];
				d_DPT[id*1331+1314]=d_DPT[id*1331+1317];
			}
		}
		else
		{
			d_DPT[id*1331+1312]=d_DPT[id*1331+1315];
			d_DPT[id*1331+1313]=d_DPT[id*1331+1316];
			d_DPT[id*1331+1314]=d_DPT[id*1331+1317];
		}
	}

	d_DPT[id*1331+1315]=parameter[5680+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]];
	d_DPT[id*1331+1316]=parameter[5705+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]];
	d_DPT[id*1331+1317]=(d_DPT[id*1331+1316]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1315]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
	if(fabs(d_DPT[id*1331+1313])<999999999)
	{
		if(d_DPT[id*1331+1314]<d_DPT[id*1331+1317])
		{
			d_DPT[id*1331+1310]=d_DPT[id*1331+1315];
			d_DPT[id*1331+1311]=d_DPT[id*1331+1316];
		}
		else
		{
			d_DPT[id*1331+1310]=d_DPT[id*1331+1312];
			d_DPT[id*1331+1311]=d_DPT[id*1331+1313];
		}
	}
	else
	{
		d_DPT[id*1331+1310]=d_DPT[id*1331+1315];
		d_DPT[id*1331+1311]=d_DPT[id*1331+1316];
	}
	return;
}

__device__ void maxTM(int i,int j,int length,double *d_DPT,char *d_numSeq,int id)
{
	d_DPT[id*1331+1314]=d_DPT[id*1331+625+(i-1)*length+j-1];
	d_DPT[id*1331+1316]=d_DPT[id*1331+(i-1)*length+j-1];
	d_DPT[id*1331+1312]=(d_DPT[id*1331+1316]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1314]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]); // at current position 
	if(fabs(d_DPT[id*1331+(i-2)*length+j-2])<999999999)
	{
		Hs(i-1,j-1,1,length,d_numSeq,id,d_DPT);
		if(fabs(d_DPT[id*1331+1330])<999999999)
		{
			Ss(i-1,j-1,1,length,d_numSeq,id,d_DPT);
			d_DPT[id*1331+1315]=(d_DPT[id*1331+625+(i-2)*length+j-2]+d_DPT[id*1331+1330]);
			Hs(i-1,j-1,1,length,d_numSeq,id,d_DPT);
			d_DPT[id*1331+1317]=(d_DPT[id*1331+(i-2)*length+j-2]+d_DPT[id*1331+1330]);
		}
	}
	else
	{
		d_DPT[id*1331+1315]=-1.0;
		d_DPT[id*1331+1317]=1.0*INFINITY;
	}
	d_DPT[id*1331+1313]=(d_DPT[id*1331+1317]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1315]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);

	if(d_DPT[id*1331+1315]<-2500.0)
	{
// to not give dH any value if dS is unreasonable
		d_DPT[id*1331+1315]=-3224.0;
		d_DPT[id*1331+1317]=0.0;
	}
	if(d_DPT[id*1331+1314]<-2500.0)
	{
// to not give dH any value if dS is unreasonable
		d_DPT[id*1331+1314]=-3224.0;
		d_DPT[id*1331+1316]=0.0;
	}
	if((d_DPT[id*1331+1313]>d_DPT[id*1331+1312])||(d_DPT[id*1331+1314]>0&&d_DPT[id*1331+1316]>0)) // T1 on suurem 
	{
		d_DPT[id*1331+625+(i-1)*length+j-1]=d_DPT[id*1331+1315];
		d_DPT[id*1331+(i-1)*length+j-1]=d_DPT[id*1331+1317];
	}
	else if(d_DPT[id*1331+1312]>=d_DPT[id*1331+1313])
	{
		d_DPT[id*1331+625+(i-1)*length+j-1]=d_DPT[id*1331+1314];
		d_DPT[id*1331+(i-1)*length+j-1]=d_DPT[id*1331+1316];
	}
}

__device__ void calc_bulge_internal(int i,int j,int ii,int jj,int traceback,int length,double *d_DPT,char *d_numSeq,int id)
{
	d_DPT[id*1331+1314]=-3224.0;
	d_DPT[id*1331+1315]=0;

	if((ii-i==1&&jj-j-1>0)||(jj-j-1==0&&ii-i-1>0))// only bulges have to be considered
	{
		if(jj-j-1==1||ii-i-1==1) // bulge loop of size one is treated differently the intervening nn-pair must be added
		{
			if((jj-j-1==1&&ii-i-1==0)||(jj-j-1==0&&ii-i-1==1))
			{
				d_DPT[id*1331+1315]=parameter[3150+ii-i-3+jj-j]+parameter[625+d_numSeq[id*54+i]*125+d_numSeq[id*54+ii]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+jj]];
				d_DPT[id*1331+1314]=parameter[3060+ii-i-3+jj-j]+parameter[d_numSeq[id*54+i]*125+d_numSeq[id*54+ii]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+jj]];
			}
			d_DPT[id*1331+1315]+=d_DPT[id*1331+(i-1)*length+j-1];
			d_DPT[id*1331+1314]+=d_DPT[id*1331+625+(i-1)*length+j-1];
			if(fabs(d_DPT[id*1331+1315])>999999999)
			{
				d_DPT[id*1331+1315]=1.0*INFINITY;
				d_DPT[id*1331+1314]=-1.0;
			}

			d_DPT[id*1331+1312]=(d_DPT[id*1331+1315]+d_DPT[id*1331+1302])/((d_DPT[id*1331+1314]+d_DPT[id*1331+1303])+d_DPT[id*1331+1304]);
			d_DPT[id*1331+1313]=(d_DPT[id*1331+(ii-1)*length+jj-1]+d_DPT[id*1331+1302])/((d_DPT[id*1331+625+(ii-1)*length+jj-1])+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
			if((d_DPT[id*1331+1312]>d_DPT[id*1331+1313])||((traceback&&d_DPT[id*1331+1312]>=d_DPT[id*1331+1313])||(traceback==1)))
			{
				d_DPT[id*1331+1310]=d_DPT[id*1331+1314];
				d_DPT[id*1331+1311]=d_DPT[id*1331+1315];
			}
		}
		else // we have _not_ implemented Jacobson-Stockaymayer equation; the maximum bulgeloop size is 30
		{
			d_DPT[id*1331+1315]=parameter[3150+ii-i-3+jj-j]+parameter[5705+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[5705+d_numSeq[id*54+ii]*5+d_numSeq[id*54+27+jj]];
			d_DPT[id*1331+1315]+=d_DPT[id*1331+(i-1)*length+j-1];

			d_DPT[id*1331+1314]=parameter[3060+ii-i-3+jj-j]+parameter[5680+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[5680+d_numSeq[id*54+ii]*5+d_numSeq[id*54+27+jj]];
			d_DPT[id*1331+1314]+=d_DPT[id*1331+625+(i-1)*length+j-1];
			if(fabs(d_DPT[id*1331+1315])>999999999)
			{
				d_DPT[id*1331+1315]=1.0*INFINITY;
				d_DPT[id*1331+1314]=-1.0;
			}
			d_DPT[id*1331+1312]=(d_DPT[id*1331+1315]+d_DPT[id*1331+1302])/((d_DPT[id*1331+1314]+d_DPT[id*1331+1303])+d_DPT[id*1331+1304]);
			d_DPT[id*1331+1313]=(d_DPT[id*1331+(ii-1)*length+jj-1]+d_DPT[id*1331+1302])/(d_DPT[id*1331+625+(ii-1)*length+jj-1]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
			if((d_DPT[id*1331+1312]>d_DPT[id*1331+1313])||((traceback&&d_DPT[id*1331+1312]>=d_DPT[id*1331+1313])||(traceback==1)))
			{
				d_DPT[id*1331+1310]=d_DPT[id*1331+1314];
				d_DPT[id*1331+1311]=d_DPT[id*1331+1315];
			}
		}
	}
	else if(ii-i-1==1&&jj-j-1==1)
	{
		d_DPT[id*1331+1314]=parameter[1250+d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j+1]]+parameter[1250+d_numSeq[id*54+27+jj]*125+d_numSeq[id*54+27+jj-1]*25+d_numSeq[id*54+ii]*5+d_numSeq[id*54+ii-1]];
		d_DPT[id*1331+1314]+=d_DPT[id*1331+625+(i-1)*length+j-1];

		d_DPT[id*1331+1315]=parameter[1875+d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j+1]]+parameter[1875+d_numSeq[id*54+27+jj]*125+d_numSeq[id*54+27+jj-1]*25+d_numSeq[id*54+ii]*5+d_numSeq[id*54+ii-1]];
		d_DPT[id*1331+1315]+=d_DPT[id*1331+(i-1)*length+j-1];
		if(fabs(d_DPT[id*1331+1315])>999999999)
		{
			d_DPT[id*1331+1315]=1.0*INFINITY;
			d_DPT[id*1331+1314]=-1.0;
		}
		d_DPT[id*1331+1312]=(d_DPT[id*1331+1315]+d_DPT[id*1331+1302])/((d_DPT[id*1331+1314]+d_DPT[id*1331+1303])+d_DPT[id*1331+1304]);
		d_DPT[id*1331+1313]=(d_DPT[id*1331+(ii-1)*length+jj-1]+d_DPT[id*1331+1302])/(d_DPT[id*1331+625+(ii-1)*length+jj-1]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
		if((d_DPT[id*1331+1312]-d_DPT[id*1331+1313]>=0.000001)||traceback==1)
		{
			if((d_DPT[id*1331+1312]>d_DPT[id*1331+1313])||(traceback&&d_DPT[id*1331+1312]>=d_DPT[id*1331+1313]))
			{
				d_DPT[id*1331+1310]=d_DPT[id*1331+1314];
				d_DPT[id*1331+1311]=d_DPT[id*1331+1315];
			}
		}
		return;
	}
	else // only internal loops
	{
		d_DPT[id*1331+1315]=parameter[3120+ii-i-3+jj-j]+parameter[3805+d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j+1]]+parameter[3805+d_numSeq[id*54+27+jj]*125+d_numSeq[id*54+27+jj-1]*25+d_numSeq[id*54+ii]*5+d_numSeq[id*54+ii-1]];
		d_DPT[id*1331+1315]+=d_DPT[id*1331+(i-1)*length+j-1];

		d_DPT[id*1331+1314]=parameter[3030+ii-i-3+jj-j]+parameter[3180+d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j+1]]+parameter[3180+d_numSeq[id*54+27+jj]*125+d_numSeq[id*54+27+jj-1]*25+d_numSeq[id*54+ii]*5+d_numSeq[id*54+ii-1]]+(-300/310.15*abs(ii-i-jj+j));
		d_DPT[id*1331+1314]+=d_DPT[id*1331+625+(i-1)*length+j-1];
		if(fabs(d_DPT[id*1331+1315])>999999999)
		{
			d_DPT[id*1331+1315]=1.0*INFINITY;
			d_DPT[id*1331+1314]=-1.0;
		}
		d_DPT[id*1331+1312]=(d_DPT[id*1331+1315]+d_DPT[id*1331+1302])/((d_DPT[id*1331+1314]+d_DPT[id*1331+1303])+d_DPT[id*1331+1304]);
		d_DPT[id*1331+1313]=(d_DPT[id*1331+(ii-1)*length+jj-1]+d_DPT[id*1331+1302])/((d_DPT[id*1331+625+(ii-1)*length+jj-1])+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
		if((d_DPT[id*1331+1312]>d_DPT[id*1331+1313])||((traceback&&d_DPT[id*1331+1312]>=d_DPT[id*1331+1313])||(traceback==1)))
		{
			d_DPT[id*1331+1310]=d_DPT[id*1331+1314];
			d_DPT[id*1331+1311]=d_DPT[id*1331+1315];
		}
	}
	return;
}

__device__ void fillMatrix(int length,double *d_DPT,char *d_numSeq,int id,int *d_ps)
{
	int i;

	for(i=1;i<=length;i++)
	{
		for(d_ps[id*119+106]=1;d_ps[id*119+106]<=length;d_ps[id*119+106]++)
		{
			d_ps[id*119+105]=i;
			if(fabs(d_DPT[id*1331+(d_ps[id*119+105]-1)*length+d_ps[id*119+106]-1])<999999999)
			{
				d_DPT[id*1331+1310]=-1.0;
				d_DPT[id*1331+1311]=1.0*INFINITY;
				LSH(d_ps[id*119+105],d_ps[id*119+106],length,d_DPT,d_numSeq,id);

				if(fabs(d_DPT[id*1331+1311])<999999999)
				{
					d_DPT[id*1331+625+(d_ps[id*119+105]-1)*length+d_ps[id*119+106]-1]=d_DPT[id*1331+1310];
					d_DPT[id*1331+(d_ps[id*119+105]-1)*length+d_ps[id*119+106]-1]=d_DPT[id*1331+1311];
				}
				if(d_ps[id*119+105]>1&&d_ps[id*119+106]>1)
				{
					maxTM(d_ps[id*119+105],d_ps[id*119+106],length,d_DPT,d_numSeq,id);
					for(d_ps[id*119+104]=3;d_ps[id*119+104]<=32;d_ps[id*119+104]++)
					{
						d_ps[id*119+108]=d_ps[id*119+106]+1-d_ps[id*119+104];
						if(d_ps[id*119+108]<1)
						{
							d_ps[id*119+107]=d_ps[id*119+105]-1+d_ps[id*119+108]-1;
							d_ps[id*119+108]=1;
						}
						else
						{
							d_ps[id*119+107]=d_ps[id*119+105]-1;
						}
						for(;d_ps[id*119+107]>0&&d_ps[id*119+108]<d_ps[id*119+106];--d_ps[id*119+107],++d_ps[id*119+108])
						{
							if(fabs(d_DPT[id*1331+(d_ps[id*119+107]-1)*length+d_ps[id*119+108]-1])<999999999)
							{
								d_DPT[id*1331+1310]=-1.0;
								d_DPT[id*1331+1311]=1.0*INFINITY;
								calc_bulge_internal(d_ps[id*119+107],d_ps[id*119+108],i,d_ps[id*119+106],0,length,d_DPT,d_numSeq,id);

								if(d_DPT[id*1331+1310]<-2500.0)
								{
									d_DPT[id*1331+1310] =-3224.0;
									d_DPT[id*1331+1311] = 0.0;
								}
								if(fabs(d_DPT[id*1331+1311])<999999999)
								{
									d_DPT[id*1331+(i-1)*length+d_ps[id*119+106]-1]=d_DPT[id*1331+1311];
									d_DPT[id*1331+625+(i-1)*length+d_ps[id*119+106]-1]=d_DPT[id*1331+1310];
								}
							}
						}
					}
				} // if 
			}
		} // for 
	} //for
}

__device__ void RSH(int i,int j,double *d_DPT,char *d_numSeq,int id)
{
	if(d_numSeq[id*54+i]+d_numSeq[id*54+27+j]!=3)
	{
		d_DPT[id*1331+1306]=-1.0;
		d_DPT[id*1331+1307]=1.0*INFINITY;
		return;
	}
	d_DPT[id*1331+1310]=parameter[5680+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[4430+d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j+1]];
	d_DPT[id*1331+1312]=parameter[5705+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[5055+d_numSeq[id*54+i]*125+d_numSeq[id*54+i+1]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j+1]];
	if(fabs(d_DPT[id*1331+1312])>999999999)
	{
		d_DPT[id*1331+1312]=1.0*INFINITY;
		d_DPT[id*1331+1310]=-1.0;
	}
	if(fabs(parameter[2625+d_numSeq[id*54+i]*25+d_numSeq[id*54+i+1]*5+d_numSeq[id*54+27+j]])<999999999&&fabs(parameter[2875+d_numSeq[id*54+i]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j+1]])<999999999)
	{
		d_DPT[id*1331+1311]=parameter[5680+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[2500+d_numSeq[id*54+i]*25+d_numSeq[id*54+i+1]*5+d_numSeq[id*54+27+j]]+parameter[2750+d_numSeq[id*54+i]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j+1]];
		d_DPT[id*1331+1313]=parameter[5705+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[2625+d_numSeq[id*54+i]*25+d_numSeq[id*54+i+1]*5+d_numSeq[id*54+27+j]]+parameter[2875+d_numSeq[id*54+i]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j+1]];
		if(fabs(d_DPT[id*1331+1313])>999999999)
		{
			d_DPT[id*1331+1313]=1.0*INFINITY;
			d_DPT[id*1331+1311]=-1.0;
		}
		d_DPT[id*1331+1315]=(d_DPT[id*1331+1313]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1311]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
		if(fabs(d_DPT[id*1331+1312])<999999999)
		{
			d_DPT[id*1331+1314]=(d_DPT[id*1331+1312]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1310]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
			if(d_DPT[id*1331+1314]<d_DPT[id*1331+1315])
			{
				d_DPT[id*1331+1310]=d_DPT[id*1331+1311];
				d_DPT[id*1331+1312]=d_DPT[id*1331+1313];
				d_DPT[id*1331+1314]=d_DPT[id*1331+1315];
			}
		}
		else
		{
			d_DPT[id*1331+1310]=d_DPT[id*1331+1311];
			d_DPT[id*1331+1312]=d_DPT[id*1331+1313];
			d_DPT[id*1331+1314]=d_DPT[id*1331+1315];
		}
	}

	if(fabs(parameter[2625+d_numSeq[id*54+i]*25+d_numSeq[id*54+i+1]*5+d_numSeq[id*54+27+j]])<999999999)
	{
		d_DPT[id*1331+1311]=parameter[5680+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[2500+d_numSeq[id*54+i]*25+d_numSeq[id*54+i+1]*5+d_numSeq[id*54+27+j]];
		d_DPT[id*1331+1313]=parameter[5705+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[2625+d_numSeq[id*54+i]*25+d_numSeq[id*54+i+1]*5+d_numSeq[id*54+27+j]];
		if(fabs(d_DPT[id*1331+1313])>999999999)
		{
			d_DPT[id*1331+1313]=1.0*INFINITY;
			d_DPT[id*1331+1311]=-1.0;
		}
		d_DPT[id*1331+1315]=(d_DPT[id*1331+1313]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1311]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
		if(fabs(d_DPT[id*1331+1312])<999999999)
		{
			d_DPT[id*1331+1314]=(d_DPT[id*1331+1312]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1310]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
			if(d_DPT[id*1331+1314]<d_DPT[id*1331+1315])
			{
				d_DPT[id*1331+1310]=d_DPT[id*1331+1311];
				d_DPT[id*1331+1312]=d_DPT[id*1331+1313];
				d_DPT[id*1331+1314]=d_DPT[id*1331+1315];
			}
		}
		else
		{
			d_DPT[id*1331+1310]=d_DPT[id*1331+1311];
			d_DPT[id*1331+1312]=d_DPT[id*1331+1313];
			d_DPT[id*1331+1314]=d_DPT[id*1331+1315];
		}
	}

	if(fabs(parameter[2875+d_numSeq[id*54+i]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j+1]])<999999999)
	{
		d_DPT[id*1331+1311]=parameter[5680+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[2750+d_numSeq[id*54+i]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j+1]];
		d_DPT[id*1331+1313]=parameter[5705+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]]+parameter[2875+d_numSeq[id*54+i]*25+d_numSeq[id*54+27+j]*5+d_numSeq[id*54+27+j+1]];
		if(fabs(d_DPT[id*1331+1313])>999999999)
		{
			d_DPT[id*1331+1313]=1.0*INFINITY;
			d_DPT[id*1331+1311]=-1.0;
		}
		d_DPT[id*1331+1315]=(d_DPT[id*1331+1313]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1311]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
		if(fabs(d_DPT[id*1331+1312])<999999999)
		{
			d_DPT[id*1331+1314]=(d_DPT[id*1331+1312]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1310]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
			if(d_DPT[id*1331+1314]<d_DPT[id*1331+1315])
			{
				d_DPT[id*1331+1310]=d_DPT[id*1331+1311];
				d_DPT[id*1331+1312]=d_DPT[id*1331+1313];
				d_DPT[id*1331+1314]=d_DPT[id*1331+1315];
			}
		}
		else
		{
			d_DPT[id*1331+1310]=d_DPT[id*1331+1311];
			d_DPT[id*1331+1312]=d_DPT[id*1331+1313];
			d_DPT[id*1331+1314]=d_DPT[id*1331+1315];
		}
	}
	d_DPT[id*1331+1311]=parameter[5680+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]];
	d_DPT[id*1331+1313]=parameter[5705+d_numSeq[id*54+i]*5+d_numSeq[id*54+27+j]];
	d_DPT[id*1331+1315]=(d_DPT[id*1331+1313]+d_DPT[id*1331+1302])/(d_DPT[id*1331+1311]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]);
	if(fabs(d_DPT[id*1331+1312])<999999999)
	{
		if(d_DPT[id*1331+1314]<d_DPT[id*1331+1315])
		{
			d_DPT[id*1331+1306]=d_DPT[id*1331+1311];
			d_DPT[id*1331+1307]=d_DPT[id*1331+1313];
		}
		else
		{
			d_DPT[id*1331+1306]=d_DPT[id*1331+1310];
			d_DPT[id*1331+1307]=d_DPT[id*1331+1312];
		}
	}
	else
	{
		d_DPT[id*1331+1306]=d_DPT[id*1331+1311];
		d_DPT[id*1331+1307]=d_DPT[id*1331+1313];
	}
	return;
}

__device__ void traceback(int i,int j,int *d_ps,int length,double *d_DPT,char *d_numSeq,int id)
{
	d_ps[id*119+i-1]=j;
	d_ps[id*119+25+j-1]=i;
	while(1)
	{
		d_DPT[id*1331+1310]=-1.0;
		d_DPT[id*1331+1311]=1.0*INFINITY;
		LSH(i,j,length,d_DPT,d_numSeq,id);
		equal(d_DPT[id*1331+625+(i-1)*length+j-1],d_DPT[id*1331+1310],d_ps,id,117);
		equal(d_DPT[id*1331+(i-1)*length+j-1],d_DPT[id*1331+1311],d_ps,id,118);
		if(d_ps[id*119+117]&&d_ps[id*119+118])
			break;

		d_ps[id*119+107] = 0;
		if(i>1&&j>1)
		{
			Ss(i-1,j-1,1,length,d_numSeq,id,d_DPT);
			equal(d_DPT[id*1331+625+(i-1)*length+j-1],d_DPT[id*1331+1330]+d_DPT[id*1331+625+(i-2)*length+j-2],d_ps,id,117);
			if(d_ps[id*119+117])
			{
				i=i-1;
				j=j-1;
				d_ps[id*119+i-1]=j;
				d_ps[id*119+25+j-1]=i;
				d_ps[id*119+107]=1;
			}
		}
		for(d_ps[id*119+104]=3;!d_ps[id*119+107]&&d_ps[id*119+104]<=32;++d_ps[id*119+104])
		{
			d_ps[id*119+105]=i-1;
			d_ps[id*119+106]=-d_ps[id*119+105]-d_ps[id*119+104]+(j+i);
			if(d_ps[id*119+106]<1)
			{
				d_ps[id*119+105]-=abs(d_ps[id*119+106]-1);
				d_ps[id*119+106]=1;
			}
			for(;!d_ps[id*119+107]&&d_ps[id*119+105]>0&&d_ps[id*119+106]<j;--d_ps[id*119+105],++d_ps[id*119+106])
			{
				d_DPT[id*1331+1310]=-1.0;
				d_DPT[id*1331+1311]=1.0*INFINITY;
				calc_bulge_internal(d_ps[id*119+105],d_ps[id*119+106],i,j,1,length,d_DPT,d_numSeq,id);
				equal(d_DPT[id*1331+625+(i-1)*length+j-1],d_DPT[id*1331+1310],d_ps,id,117);
				equal(d_DPT[id*1331+(i-1)*length+j-1],d_DPT[id*1331+1311],d_ps,id,118);
				if(d_ps[id*119+117]&&d_ps[id*119+118])
				{
					i=d_ps[id*119+105];
					j=d_ps[id*119+106];
					d_ps[id*119+i-1]=j;
					d_ps[id*119+25+j-1]=i;
					d_ps[id*119+107]=1;
					break;
				}
			}
		}
	}
}

__device__ void drawDimer(int *d_ps,int id,double H,double S,double *d_DPT,int length)
{
        if(fabs(d_DPT[id*1331+1305])>999999999)
                d_DPT[id*1331+1309]=0.0;
        else
        {
                d_ps[id*119+105]=0;
                for(d_ps[id*119+104]=0;d_ps[id*119+104]<length;d_ps[id*119+104]++)
                {
                        if(d_ps[id*119+d_ps[id*119+104]]>0)
                                ++d_ps[id*119+105];
                }
                for(d_ps[id*119+104]=0;d_ps[id*119+104]<length;d_ps[id*119+104]++)
                {
                        if(d_ps[id*119+25+d_ps[id*119+104]]>0)
                                ++d_ps[id*119+105];
                }
                d_ps[id*119+105]=(d_ps[id*119+105]/2)-1;
                d_DPT[id*1331+1309]=(H/(S+(d_ps[id*119+105]*-0.51986)+d_DPT[id*1331+1304])-273.15);
        }
}

__device__ void symmetry_thermo(char *d_seq,int start,int length,int id, int *d_ps)
{
	if(length%2==1)
	{
		d_ps[id*119+101]=0;
		return;
	}
	d_ps[id*119+100]=0;
	while(d_ps[id*119+100]<length/2)
	{
		if((d_seq[start+d_ps[id*119+100]]=='A'&&d_seq[start+length-1-d_ps[id*119+100]]!='T')||(d_seq[start+d_ps[id*119+100]]=='T'&&d_seq[start+length-1-d_ps[id*119+100]]!='A')||(d_seq[start+length-1-d_ps[id*119+100]]=='A'&&d_seq[start+d_ps[id*119+100]]!='T')||(d_seq[start+length-1-d_ps[id*119+100]]=='T'&&d_seq[start+d_ps[id*119+100]]!='A'))
		{
			d_ps[id*119+101]=0;
			return;
		}
		if((d_seq[start+d_ps[id*119+100]]=='C'&&d_seq[start+length-1-d_ps[id*119+100]]!='G')||(d_seq[start+d_ps[id*119+100]]=='G'&&d_seq[start+length-1-d_ps[id*119+100]]!='C')||(d_seq[start+length-1-d_ps[id*119+100]]=='C'&&d_seq[start+d_ps[id*119+100]]!='G')||(d_seq[start+length-1-d_ps[id*119+100]]=='G'&&d_seq[start+d_ps[id*119+100]]!='C'))
		{
			d_ps[id*119+101]=0;
			return;
		}
		d_ps[id*119+100]++;
	}
	d_ps[id*119+101]=1;
}

__device__ void thal(char *d_seq,int start,int length,int strand_flag,int type,char *d_numSeq,int id,double *d_DPT,int *d_ps)
{
	if (type==4) /* unimolecular folding */
	{
		d_DPT[id*1331+1302]= 0.0;
		d_DPT[id*1331+1303] = -0.00000000001;
		d_DPT[id*1331+1304]=0;
	}
	else /* hybridization of two oligos */
	{
		d_DPT[id*1331+1302]= 200;
		d_DPT[id*1331+1303]= -5.7;
		symmetry_thermo(d_seq,start,length,id,d_ps);
		if(d_ps[id*119+101]==1)
			d_DPT[id*1331+1304]=1.9872* log(38/1000000000.0);
		else
			d_DPT[id*1331+1304]=1.9872* log(38/4000000000.0);
	}
/* convert nucleotides to numbers */
	if(type==1 || type==2)
	{
		if(strand_flag==0) //plus
		{
	 		for(d_ps[id*119+102]=1;d_ps[id*119+102]<=length;++d_ps[id*119+102])
			{
				str2int(d_seq[start+d_ps[id*119+102]-1],d_numSeq,(id*54+d_ps[id*119+102]));
				str2int(d_seq[start+length-d_ps[id*119+102]],d_numSeq,(id*54+27+d_ps[id*119+102]));
			}
		}
		else
		{
			for(d_ps[id*119+102]=1;d_ps[id*119+102]<=length;++d_ps[id*119+102])
			{
				str2int_rev(d_seq[start+length-d_ps[id*119+102]],d_numSeq,(id*54+d_ps[id*119+102]));
				str2int_rev(d_seq[start+d_ps[id*119+102]-1],d_numSeq,(id*54+27+d_ps[id*119+102]));
			}
		}
	}
	else
	{
		if(strand_flag==0)
		{
                	for(d_ps[id*119+102]=1;d_ps[id*119+102]<=length;++d_ps[id*119+102])
			{
				str2int(d_seq[start+d_ps[id*119+102]-1],d_numSeq,(id*54+d_ps[id*119+102]));
				d_numSeq[id*54+27+d_ps[id*119+102]]=d_numSeq[id*54+d_ps[id*119+102]];
			}
		}
		else
		{
			for(d_ps[id*119+102]=1;d_ps[id*119+102]<=length;++d_ps[id*119+102])
			{
				str2int_rev(d_seq[start+length-d_ps[id*119+102]],d_numSeq,(id*54+d_ps[id*119+102]));
				d_numSeq[id*54+27+d_ps[id*119+102]]=d_numSeq[id*54+d_ps[id*119+102]];
			}
		}
	}
	d_numSeq[id*54+0]=d_numSeq[id*54+length+1]=d_numSeq[id*54+27+0]=d_numSeq[id*54+27+length+1]=4; /* mark as N-s */

	d_DPT[id*1331+1309]=0;
	if (type==4) /* calculate structure of monomer */
	{
		initMatrix2(length,d_DPT,d_numSeq,id,d_ps);
		fillMatrix2(length,d_DPT,d_numSeq,id,d_ps);
		calc_terminal_bp(310.15,length,d_DPT,d_numSeq,id,d_ps);
		d_DPT[id*1331+1306]=d_DPT[id*1331+1276+length];
		d_DPT[id*1331+1307]=d_DPT[id*1331+1250+length];
		for (d_ps[id*119+102]=0;d_ps[id*119+102]<length;d_ps[id*119+102]++)
			d_ps[id*119+d_ps[id*119+102]]=0;
		if(fabs(d_DPT[id*1331+1306])<999999999)
		{
			tracebacku(d_ps,length,d_DPT,d_numSeq,id);
			drawHairpin(d_ps,id,d_DPT[id*1331+1306],d_DPT[id*1331+1307],length,d_DPT);
			d_DPT[id*1331+1309]=(int)(d_DPT[id*1331+1309]*100+0.5)/100.0;
		}
	}
	else  /* Hybridization of two moleculs */
	{
		initMatrix(length,d_DPT,d_numSeq,id,d_ps);
		fillMatrix(length,d_DPT,d_numSeq,id,d_ps);

		d_DPT[id*1331+1305]=-1.0*INFINITY;
	/* calculate terminal basepairs */
		d_ps[id*119+100]=d_ps[id*119+101]=0;
		if(type==1)
			for (d_ps[id*119+102]=1;d_ps[id*119+102]<=length;d_ps[id*119+102]++)
			{
				for (d_ps[id*119+103]=1;d_ps[id*119+103]<=length;d_ps[id*119+103]++)
				{
					RSH(d_ps[id*119+102],d_ps[id*119+103],d_DPT,d_numSeq,id);
					d_DPT[id*1331+1306]=d_DPT[id*1331+1306]+0.000001; /* this adding is done for compiler, optimization -O2 vs -O0 */
					d_DPT[id*1331+1307]=d_DPT[id*1331+1307]+0.000001;
					d_DPT[id*1331+1308]=((d_DPT[id*1331+(d_ps[id*119+102]-1)*length+d_ps[id*119+103]-1]+d_DPT[id*1331+1307]+d_DPT[id*1331+1302]) / ((d_DPT[id*1331+625+(d_ps[id*119+102]-1)*length+d_ps[id*119+103]-1])+d_DPT[id*1331+1306]+d_DPT[id*1331+1303] + d_DPT[id*1331+1304])) -273.15;
					if(d_DPT[id*1331+1308]>d_DPT[id*1331+1305]&&((d_DPT[id*1331+625+(d_ps[id*119+102]-1)*length+d_ps[id*119+103]-1]+d_DPT[id*1331+1306])<0&&(d_DPT[id*1331+1307]+d_DPT[id*1331+(d_ps[id*119+102]-1)*length+d_ps[id*119+103]-1])<0))
					{
						d_DPT[id*1331+1305]=d_DPT[id*1331+1308];
						d_ps[id*119+100]=d_ps[id*119+102];
						d_ps[id*119+101]=d_ps[id*119+103];
					}
				}
			}
		if(type==2)
		{
		 //THAL_END1
			d_ps[id*119+101]=0;
			d_ps[id*119+100]=length;
			d_DPT[id*1331+1305]=-1.0*INFINITY;
			for (d_ps[id*119+103]=1;d_ps[id*119+103]<=length;++d_ps[id*119+103])
			{
				RSH(length,d_ps[id*119+103],d_DPT,d_numSeq,id);
				d_DPT[id*1331+1306]=d_DPT[id*1331+1306]+0.000001; // this adding is done for compiler, optimization -O2 vs -O0,that compiler could understand that SH is changed in this cycle 
				d_DPT[id*1331+1307]=d_DPT[id*1331+1307]+0.000001;
				d_DPT[id*1331+1308]=((d_DPT[id*1331+(length-1)*length+d_ps[id*119+103]-1]+d_DPT[id*1331+1307]+d_DPT[id*1331+1302])/((d_DPT[id*1331+625+(length-1)*length+d_ps[id*119+103]-1])+d_DPT[id*1331+1306]+d_DPT[id*1331+1303]+d_DPT[id*1331+1304]))-273.15;
				if (d_DPT[id*1331+1308]>d_DPT[id*1331+1305]&&((d_DPT[id*1331+1306]+d_DPT[id*1331+625+(length-1)*length+d_ps[id*119+103]-1])<0&&(d_DPT[id*1331+1307]+d_DPT[id*1331+(length-1)*length+d_ps[id*119+103]-1])<0))
				{
					d_DPT[id*1331+1305]=d_DPT[id*1331+1308];
					d_ps[id*119+101]=d_ps[id*119+103];
				}
			}
		}
		if(fabs(d_DPT[id*1331+1305])>999999999)
			d_ps[id*119+100]=d_ps[id*119+101]=1;
		RSH(d_ps[id*119+100],d_ps[id*119+101],d_DPT,d_numSeq,id);
	 // tracebacking
		for (d_ps[id*119+102]=0;d_ps[id*119+102]<length;++d_ps[id*119+102])
			d_ps[id*119+d_ps[id*119+102]]=0;
		for (d_ps[id*119+103]=0;d_ps[id*119+103]<length;++d_ps[id*119+103])
			d_ps[id*119+25+d_ps[id*119+103]] = 0;
		if(fabs(d_DPT[id*1331+(d_ps[id*119+100]-1)*length+d_ps[id*119+101]-1])<999999999)
		{
			traceback(d_ps[id*119+100],d_ps[id*119+101],d_ps,length,d_DPT,d_numSeq,id);
			drawDimer(d_ps,id,(d_DPT[id*1331+(d_ps[id*119+100]-1)*length+d_ps[id*119+101]-1]+d_DPT[id*1331+1307]+d_DPT[id*1331+1302]),(d_DPT[id*1331+625+(d_ps[id*119+100]-1)*length+d_ps[id*119+101]-1]+d_DPT[id*1331+1306]+d_DPT[id*1331+1303]),d_DPT,length);
			d_DPT[id*1331+1309]=(int)(d_DPT[id*1331+1309]*100+0.5)/100.0;
		}
	}
}

///function in gpu, check the GC-content; int length: the length of read
__device__ float gc(char *d_seq,int start,int length)
{
	int i,number;

	number=0;
	for(i=0;i<length;i++)
	{
		if(d_seq[start+i]=='C')
		{
			number++;
			continue;
		}
	
		if(d_seq[start+i]=='G')
		{
			number++;
		}
	}

	return 1.0*number/length*100;
}

///function in gpu, translate A...G to int
__device__ int translate(char a)
{
	if(a=='A')
		return 0;
	if(a=='T')
		return 1;
	if(a=='C')
		return 2;
	return 3;
}

__device__ int translate_rev(char a)
{
        if(a=='T')
                return 0;
        if(a=='A')
                return 1;
        if(a=='G')
                return 2;
        return 3;
}
//function in gpu, caculate tm
__device__ float tm(char *d_seq,int start,int length)
{
	int i,pos;
	float deltah,deltas;

	deltah=0;
	deltas=0;
	for(i=0;i<length-1;i++)
	{
		pos=translate(d_seq[start+i]);
		pos=pos*4+translate(d_seq[start+i+1]);
		deltah+=d_deltah[pos];
		deltas+=d_deltas[pos];
	}

	deltah=(-1.0)*deltah;
	deltas=(-1.0)*deltas;
	if((d_seq[start]=='A')||(d_seq[start]=='T'))
	{
		deltah+=2.3;
		deltas+=4.1;
	}
	else
	{
		deltah+=0.1;
		deltas-=2.8;
	}
        if((d_seq[start+length-1]=='A')||(d_seq[start+length-1]=='T'))
        {
                deltah+=2.3;
                deltas+=4.1;
        }
        else
        {
                deltah+=0.1;
                deltas-=2.8;
        }
	return 1000.0*deltah/(deltas-0.51986*(length-1)-36.70381)-273.15;
}

///function in gpu, caculate stability, int strand: 0 is 5' and 1 is 3'
__device__ void stability(char *d_seq,int start,int flag,int length,float Stab[],int id)//flag=0: plus
{
	int i,pos;
	
	pos=0;
	for(i=0;i<6;i++)
	{
		if(flag==0)
			pos=pos*4+translate(d_seq[start+i]);
		else
			pos=pos*4+translate_rev(d_seq[start+length-1-i]);
	}
	Stab[2*id]=d_stab[pos];

//3'
        pos=0;
        for(i=0;i<6;i++)
        {
		if(flag==0)
			pos=pos*4+translate(d_seq[start+i+length-6]);
		else //minus
			pos=pos*4+translate_rev(d_seq[start+5-i]);
        }
	Stab[2*id+1]=d_stab[pos];
	return;
}

__device__ int dimer(char *d_seq,int start,int length) //length=6: 5'; else :3'
{
//same
	if(length==6)
	{
		if(d_seq[start]==d_seq[start+1]&&d_seq[start]==d_seq[start+2]&&d_seq[start]==d_seq[start+3])
			return 0;
	}
	else
	{
        	if(d_seq[start+length-1]==d_seq[start+length-2]&&d_seq[start+length-1]==d_seq[start+length-3]&&d_seq[start+length-1]==d_seq[start+length-4])
        	        return 0;
	}
        if(d_seq[start+length-1]=='A')
        {
                if(d_seq[start+length-6]!='T')
                        return 1;
        }
        else if(d_seq[start+length-1]=='T')
        {
                if(d_seq[start+length-6]!='A')
                        return 1;
        }
        else if(d_seq[start+length-1]=='C')
        {
                if(d_seq[start+length-6]!='G')
                        return 1;
        }
        else
        {
                if(d_seq[start+length-1]!='C')
                        return 1;
        }

        if(d_seq[start+length-2]=='A')
        {        
                if(d_seq[start+length-5]!='T')
                        return 1;        
        }
        else if(d_seq[start+length-2]=='T')
        {        
                if(d_seq[start+length-5]!='A')
                        return 1;
        }                
        else if(d_seq[start+length-2]=='C')
        {
                if(d_seq[start+length-5]!='G')
                        return 1;  
        }
        else
        {
                if(d_seq[start+length-5]!='C')   
                        return 1;  
        }

        if(d_seq[start+length-3]=='A')
        {        
                if(d_seq[start+length-4]!='T')
                        return 1;        
        }
        else if(d_seq[start+length-3]=='T')
        {        
                if(d_seq[start+length-4]!='A')
                        return 1;
        }                
        else if(d_seq[start+length-3]=='C')
        {
                if(d_seq[start+length-4]!='G')
                        return 1;  
        }
        else
        {
                if(d_seq[start+length-4]!='C')   
                        return 1;  
        }
        return 0;
}


//function in gpu: whether species chars in reads
__device__ int words(char *d_seq,int position,int length)
{
	int i;
	
	for(i=0;i<length;i++)
	{
		if(d_seq[position+i]=='N')
		{
			return 0;
		}
	}
	return 1;
}
/*
__device__ int check_long_ploy(char *d_seq,int start,int length)
{
        int i,same;
        char ref;

        same=1;
        ref=d_seq[start];
        for(i=1;i<length;i++)
        {
                if(d_seq[start+i]==ref)
                        same++;
                else
                {
                        if(same>=6)
                                return 0;
                        same=1;
                        ref=d_seq[start+i];
                }
        }
        if(same>=6)
                return 0;
        return 1;
}
*/
///function: int length: the length of genome
__global__ void candidate_primer(char *d_seq,int *d_len,int *d_rev_len,int loop_flag,int length,int check_flag,char *d_numSeq,double *d_DPT,int *d_ps,float *d_Tm)
{
	__shared__ float GC[512];
	__shared__ float Tm[512];
	__shared__ float Stab[1024];
	__shared__ int pos[512];
	__shared__ int Len[512];
	__shared__ int plus[512];
	__shared__ int minus[512];

	for(pos[threadIdx.x]=threadIdx.x+blockIdx.x*blockDim.x;pos[threadIdx.x]<length;pos[threadIdx.x]=pos[threadIdx.x]+blockDim.x*gridDim.x)
	{
		for(Len[threadIdx.x]=0;Len[threadIdx.x]<11;Len[threadIdx.x]++)   //primer length is from 18 to 25
		{
			d_len[11*pos[threadIdx.x]+Len[threadIdx.x]]=0;
			d_rev_len[11*pos[threadIdx.x]+Len[threadIdx.x]]=0;
		}
	
		for(Len[threadIdx.x]=15;Len[threadIdx.x]<=25;Len[threadIdx.x]++)  //read length is from 18 to 25
		{
			if(pos[threadIdx.x]+Len[threadIdx.x]>length)
				break;
			if(words(d_seq,pos[threadIdx.x],Len[threadIdx.x])==0)
                                break;

		//	test[threadIdx.x*6+3]=check_long_ploy(d_seq,pos[threadIdx.x],Len[threadIdx.x]);
		//	if(test[threadIdx.x*6+3]==0)
		//		break;
			GC[threadIdx.x]=gc(d_seq,pos[threadIdx.x],Len[threadIdx.x]);
			if(GC[threadIdx.x]<30||GC[threadIdx.x]>70)
				continue;

			Tm[threadIdx.x]=tm(d_seq,pos[threadIdx.x],Len[threadIdx.x]);
			if(Tm[threadIdx.x]<55||Tm[threadIdx.x]>68)
				continue;

			plus[threadIdx.x]=dimer(d_seq,pos[threadIdx.x],Len[threadIdx.x]);
			minus[threadIdx.x]=dimer(d_seq,pos[threadIdx.x],6);

		//secondary structure
			if(check_flag&&plus[threadIdx.x])
			{
				thal(d_seq,pos[threadIdx.x],Len[threadIdx.x],0,1,d_numSeq,(threadIdx.x+blockIdx.x*blockDim.x),d_DPT,d_ps);
				if(d_DPT[(threadIdx.x+blockIdx.x*blockDim.x)*1331+1309]>Tm[threadIdx.x]-10)
					plus[threadIdx.x]=0;	
			}
			if(check_flag&&plus[threadIdx.x])
                        {
                                thal(d_seq,pos[threadIdx.x],Len[threadIdx.x],0,2,d_numSeq,(threadIdx.x+blockIdx.x*blockDim.x),d_DPT,d_ps);
				if(d_DPT[(threadIdx.x+blockIdx.x*blockDim.x)*1331+1309]>Tm[threadIdx.x]-10)  
                                        plus[threadIdx.x]=0;
                        }
			if(check_flag&&plus[threadIdx.x])
                        {
                                thal(d_seq,pos[threadIdx.x],Len[threadIdx.x],0,4,d_numSeq,(threadIdx.x+blockIdx.x*blockDim.x),d_DPT,d_ps);
				if(d_DPT[(threadIdx.x+blockIdx.x*blockDim.x)*1331+1309]>Tm[threadIdx.x]-10)
                                        plus[threadIdx.x]=0;
                        }
			if(plus[threadIdx.x])
			{
				stability(d_seq,pos[threadIdx.x],0,Len[threadIdx.x],Stab,threadIdx.x);
			//inner
                                if(Stab[2*threadIdx.x]>=4&&Stab[2*threadIdx.x+1]>=3&&Len[threadIdx.x]<=22&&Tm[threadIdx.x]>=64&&GC[threadIdx.x]>=40)
                                        d_len[pos[threadIdx.x]*11+Len[threadIdx.x]-15]++; //GC-rich
                                if(Stab[2*threadIdx.x]>=4&&Stab[2*threadIdx.x+1]>=3&&Len[threadIdx.x]>=20&&Tm[threadIdx.x]>=60&&Tm[threadIdx.x]<=63&&GC[threadIdx.x]<=65)
                                        d_len[pos[threadIdx.x]*11+Len[threadIdx.x]-15]+=2; //AT-rich
                                if(Stab[2*threadIdx.x]>=4&&Stab[2*threadIdx.x+1]>=3&&Len[threadIdx.x]>=20&&Len[threadIdx.x]<=22&&Tm[threadIdx.x]>=64&&Tm[threadIdx.x]<=66&&GC[threadIdx.x]>=40&&GC[threadIdx.x]<=65)
                                        d_len[pos[threadIdx.x]*11+Len[threadIdx.x]-15]+=4;

                        //outer
                                if(Stab[2*threadIdx.x]>=3&&Stab[2*threadIdx.x+1]>=4&&Len[threadIdx.x]<=20&&Tm[threadIdx.x]>=59&&Tm[threadIdx.x]<=63&&GC[threadIdx.x]>=40)
                                        d_len[pos[threadIdx.x]*11+Len[threadIdx.x]-15]+=10;
                                if(Stab[2*threadIdx.x]>=3&&Stab[2*threadIdx.x+1]>=4&&Len[threadIdx.x]>=18&&Tm[threadIdx.x]<=58&&GC[threadIdx.x]<=65)
                                        d_len[pos[threadIdx.x]*11+Len[threadIdx.x]-15]+=20;
                                if(Stab[2*threadIdx.x]>=3&&Stab[2*threadIdx.x+1]>=4&&Len[threadIdx.x]>=18&&Len[threadIdx.x]<=20&&Tm[threadIdx.x]>=59&&Tm[threadIdx.x]<=61&&GC[threadIdx.x]>=40&&GC[threadIdx.x]<=65)
                                        d_len[pos[threadIdx.x]*11+Len[threadIdx.x]-15]+=40;
                        //loop
                                if(loop_flag&&Stab[2*threadIdx.x]>=3&&Stab[2*threadIdx.x+1]>=4&&Len[threadIdx.x]<=22&&Tm[threadIdx.x]>=64&&Tm[threadIdx.x]<=68&&GC[threadIdx.x]>=40)
                                        d_len[pos[threadIdx.x]*11+Len[threadIdx.x]-15]+=100; //GC-rich
                                if(loop_flag&&Stab[2*threadIdx.x]>=3&&Stab[2*threadIdx.x+1]>=4&&Len[threadIdx.x]>=20&&Tm[threadIdx.x]>=60&&Tm[threadIdx.x]<=63&&GC[threadIdx.x]<=65)
                                        d_len[pos[threadIdx.x]*11+Len[threadIdx.x]-15]+=200; //AT-rich
                                if(loop_flag&&Stab[2*threadIdx.x]>=3&&Stab[2*threadIdx.x+1]>=4&&Len[threadIdx.x]>=20&&Len[threadIdx.x]<=22&&Tm[threadIdx.x]>=64&&Tm[threadIdx.x]<=66&&GC[threadIdx.x]>=40&&GC[threadIdx.x]<=65)
                                        d_len[pos[threadIdx.x]*11+Len[threadIdx.x]-15]+=400;
			}
	//reverse
                        if(check_flag&&minus[threadIdx.x])
                        {                
                                thal(d_seq,pos[threadIdx.x],Len[threadIdx.x],1,1,d_numSeq,(threadIdx.x+blockIdx.x*blockDim.x),d_DPT,d_ps);
				if(d_DPT[(threadIdx.x+blockIdx.x*blockDim.x)*1331+1309]>Tm[threadIdx.x]-10)
                                        minus[threadIdx.x]=0;
                        }           
                        if(check_flag&&minus[threadIdx.x])
                        {
                                thal(d_seq,pos[threadIdx.x],Len[threadIdx.x],1,2,d_numSeq,(threadIdx.x+blockIdx.x*blockDim.x),d_DPT,d_ps);
				if(d_DPT[(threadIdx.x+blockIdx.x*blockDim.x)*1331+1309]>Tm[threadIdx.x]-10)
                                        minus[threadIdx.x]=0;
                        }                
                        if(check_flag&&minus[threadIdx.x])
                        {
                                thal(d_seq,pos[threadIdx.x],Len[threadIdx.x],1,4,d_numSeq,(threadIdx.x+blockIdx.x*blockDim.x),d_DPT,d_ps);
				if(d_DPT[(threadIdx.x+blockIdx.x*blockDim.x)*1331+1309]>Tm[threadIdx.x]-10)
                                        minus[threadIdx.x]=0;
                        }
			if(minus[threadIdx.x])
                        {
                                stability(d_seq,pos[threadIdx.x],1,Len[threadIdx.x],Stab,threadIdx.x);
				//inner
                                if(Stab[2*threadIdx.x]>=4&&Stab[2*threadIdx.x+1]>=3&&Len[threadIdx.x]<=22&&Tm[threadIdx.x]>=64&&GC[threadIdx.x]>=40)        
                                        d_rev_len[pos[threadIdx.x]*11+Len[threadIdx.x]-15]++; //GC-rich
                                if(Stab[2*threadIdx.x]>=4&&Stab[2*threadIdx.x+1]>=3&&Len[threadIdx.x]>=20&&Tm[threadIdx.x]>=60&&Tm[threadIdx.x]<=63&&GC[threadIdx.x]<=65)       
                                        d_rev_len[pos[threadIdx.x]*11+Len[threadIdx.x]-15]+=2; //AT-rich
                                if(Stab[2*threadIdx.x]>=4&&Stab[2*threadIdx.x+1]>=3&&Len[threadIdx.x]>=20&&Len[threadIdx.x]<=22&&Tm[threadIdx.x]>=64&&Tm[threadIdx.x]<=66&&GC[threadIdx.x]>=40&&GC[threadIdx.x]<=65)
                                        d_rev_len[pos[threadIdx.x]*11+Len[threadIdx.x]-15]+=4;

                        //outer
                                if(Stab[2*threadIdx.x]>=3&&Stab[2*threadIdx.x+1]>=4&&Len[threadIdx.x]<=20&&Tm[threadIdx.x]>=59&&Tm[threadIdx.x]<=63&&GC[threadIdx.x]>=40)
                                        d_rev_len[pos[threadIdx.x]*11+Len[threadIdx.x]-15]+=10;
                                if(Stab[2*threadIdx.x]>=3&&Stab[2*threadIdx.x+1]>=4&&Len[threadIdx.x]>=18&&Tm[threadIdx.x]<=58&&GC[threadIdx.x]<=65)
                                        d_rev_len[pos[threadIdx.x]*11+Len[threadIdx.x]-15]+=20;
                                if(Stab[2*threadIdx.x]>=3&&Stab[2*threadIdx.x+1]>=4&&Len[threadIdx.x]>=18&&Len[threadIdx.x]<=20&&Tm[threadIdx.x]>=59&&Tm[threadIdx.x]<=61&&GC[threadIdx.x]>=40&&GC[threadIdx.x]<=65)
                                        d_rev_len[pos[threadIdx.x]*11+Len[threadIdx.x]-15]+=40;
                        //loop
                                if(loop_flag&&Stab[2*threadIdx.x]>=3&&Stab[2*threadIdx.x+1]>=4&&Len[threadIdx.x]<=22&&Tm[threadIdx.x]>=64&&GC[threadIdx.x]>=40)        
                                        d_rev_len[pos[threadIdx.x]*11+Len[threadIdx.x]-15]+=100; //GC-rich
                                if(loop_flag&&Stab[2*threadIdx.x]>=3&&Stab[2*threadIdx.x+1]>=4&&Len[threadIdx.x]>=20&&Tm[threadIdx.x]>=60&&Tm[threadIdx.x]<=63&&GC[threadIdx.x]<=65)       
                                        d_rev_len[pos[threadIdx.x]*11+Len[threadIdx.x]-15]+=200; //AT-rich
                                if(loop_flag&&Stab[2*threadIdx.x]>=3&&Stab[2*threadIdx.x+1]>=4&&Len[threadIdx.x]>=20&&Len[threadIdx.x]<=22&&Tm[threadIdx.x]>=64&&Tm[threadIdx.x]<=66&&GC[threadIdx.x]>=40&&GC[threadIdx.x]<=65)
                                        d_rev_len[pos[threadIdx.x]*11+Len[threadIdx.x]-15]+=400;
                        }
			d_Tm[pos[threadIdx.x]*11+Len[threadIdx.x]-15]=Tm[threadIdx.x];
		}
	}
	__syncthreads();
}

void usage()
{
	printf("USAGE:\n");
        printf("  Single  -in <ref_genome>  -out <single_primers>  [options]*\n\n");
        printf("ARGUMENTS:\n");
        printf("  -in <ref_genome>\n");
        printf("    reference genome, fasta formate\n");
        printf("  -out <single_primers>\n");
        printf("    output the candidate single primers\n");
        printf("  -dir <directory>\n");
        printf("    the directory for output file\n");
        printf("    default: current directory\n");
        printf("  -loop\n");
        printf("    identifiy candidate single primer regions for loop primers\n");
        printf("  -check <int>\n");
        printf("    check single primers' secondary structure or not\n");
        printf("    0: don't check secondary structure; other values: check\n");
        printf("    default: 1\n");
        printf("  -par <par_directory>\n");
        printf("    parameter files under the directory are used to check primers' secondary structure\n");
        printf("    default: GLAPD/Par/\n");
        printf("  -h[-help]\n");
        printf("    print usage\n");
}

void create_file(char *prefix,char *dir,char *seq,int *len,int *rev_len,int length,int loop_flag,int Num[],float *h_Tm)
{
	char *file;
	int plus,minus,i,j;
	FILE *Inner,*Outer,*Loop;

	i=strlen(dir)+strlen(prefix)+20;
	file=(char *)malloc(i);
//Inner
        memset(file,'\0',i);
	strcpy(file,dir);
	strcat(file,"Inner/");
	mkdir(file,0755);
	strcat(file,prefix);
	Inner=fopen(file,"w");
        if(Inner==NULL)
        {
                printf("Error! Can't create the %s file!\n",file);
                exit(1);
        }
//Outer
	memset(file,'\0',i);
        strcpy(file,dir);
        strcat(file,"Outer/");
        mkdir(file,0755);
        strcat(file,prefix);
        Outer=fopen(file,"w");
        if(Outer==NULL)
        {
                printf("Error! Can't create the %s file!\n",file);
                exit(1);
        }
//Loop
	if(loop_flag)
	{
		memset(file,'\0',i);
		strcpy(file,dir);    
		strcat(file,"Loop/");
		mkdir(file,0755);    
		strcat(file,prefix);     
		Loop=fopen(file,"w");  
		if(Loop==NULL) 
        	{
                	printf("Error! Can't create the %s file!\n",file);
                	exit(1);
        	}
	}
        for(i=0;i<length;i++)
        {
                for(j=0;j<11;j++)
                {
                        if((len[11*i+j]+rev_len[11*i+j])==0)
                                continue;
		//Inner
			plus=len[11*i+j]%10;
			minus=rev_len[11*i+j]%10;
			if(plus||minus)
			{
                       		fprintf(Inner,"pos:%d\tlength:%d\t+:%d\t-:%d\t%0.2f\n",i,(j+15),plus,minus,h_Tm[11*i+j]);
				Num[0]++;
			}
		//Outer
			plus=len[11*i+j]/10;
			plus=plus%10;
			minus=rev_len[11*i+j]/10;
			minus=minus%10;
			if(plus||minus)
			{
				fprintf(Outer,"pos:%d\tlength:%d\t+:%d\t-:%d\t%0.2f\n",i,(j+15),plus,minus,h_Tm[11*i+j]);
				Num[1]++;
			}
		//Loop
			if(loop_flag==0)
				continue;
			plus=len[11*i+j]/100;
                        minus=rev_len[11*i+j]/100;
                        if(plus||minus)
                        {
                                fprintf(Loop,"pos:%d\tlength:%d\t+:%d\t-:%d\t%0.2f\n",i,(j+15),plus,minus,h_Tm[11*i+j]);
                                Num[2]++;
                        }
                }
        }
	fclose(Inner);
	fclose(Outer);
	if(loop_flag)
		fclose(Loop);
	free(file);
}

int main(int argc, char **argv)
{
	double *H_parameter,*d_DPT;
	int *len,*d_len,length,flag[10],i,*rev_len,*d_rev_len,Num[3],NumL[2],thread,block,*d_ps;
	float deltah[16],deltas[16],stab[4096],temp1,temp2,*d_Tm,*h_Tm;
	char *seq,*d_seq,*store_path,*prefix,*stab_path,*tm_path,*curren_path,*input,*par_path,*temp,*Pchar,*d_numSeq;
	FILE *fp;
	time_t start,end;
        struct stat statbuf;
//flag: 0:input; 1: out_prefix; 2: dir; 3: stab; 4: tm; 5: high; 6: low; 7: loop; 8: secondary structure; 9: path for secondary structure

	start=time(NULL);
	thread=256;
	block=200;
//get input
        for(i=0;i<10;i++)
        {
                flag[i]=0;
        }
	flag[8]=1;
        for(i=1;i<argc;)
        {
                if(strcmp(argv[i],"-in")==0)
                {
                        flag[0]=1;
                        if(i+1==argc)
                        {
                                printf("Error! The \"-in\" parameter is not completed.\n");
                                usage();
                                exit(1);
                        }
			length=strlen(argv[i+1]);
			input=(char *)malloc(length+1);
			memset(input,'\0',length+1);
                        strcpy(input,argv[i+1]);
                        i=i+2;
                }
                else if(strcmp(argv[i],"-out")==0)
                {
                        flag[1]=1;
                        if(i+1==argc)
                        {
                                printf("Error! The \"-out\" parameter is not completed.\n");
                                usage();
                                exit(1);
                        }
			length=strlen(argv[i+1]);
                        prefix=(char *)malloc(length+1);
                        memset(prefix,'\0',length+1);
                        strcpy(prefix,argv[i+1]);
                        i=i+2;
                }
                else if(strcmp(argv[i],"-dir")==0)
                {
                        flag[2]=1;
                        if(i+1==argc)
                        {
                                printf("Error! The \"-dir\" parameter is not completed.\n");
                                usage();
                                exit(1);
                        }
			length=strlen(argv[i+1]);
			if(argv[i+1][length-1]=='/')
			{
                        	store_path=(char *)malloc(length+1);
                        	memset(store_path,'\0',length+1);
                        	strcpy(store_path,argv[i+1]);
			}
			else
			{
				store_path=(char *)malloc(length+2);
				memset(store_path,'\0',length+2);
				strcpy(store_path,argv[i+1]);
				store_path[length]='/';
			}
                        i=i+2;
                }
                else if(strcmp(argv[i],"-loop")==0) 
                {
                        flag[7]=1;
                        i++;
                }
                else if(strcmp(argv[i],"-h")==0 || strcmp(argv[i],"-help")==0)
                {
                        usage();
                        exit(1);
                }
		else if(strcmp(argv[i],"-check")==0)
                {
                        if(i+1==argc)
                        {
                                printf("Error! The \"-check\" parameter is not completed.\n");
                                usage();
                                exit(1);
                        }
                        flag[8]=atoi(argv[i+1]);
                        i=i+2;
                }
                else if(strcmp(argv[i],"-par")==0)
                {
                        flag[9]=1;
                        if(i+1==argc)
                        {
                                printf("Error! The \"-par\" parameter is not completed.\n");
                                usage();
                                exit(1);
                        }
                        length=strlen(argv[i+1]);
                        if(argv[i+1][length-1]=='/')
                        {
                                par_path=(char *)malloc(length+1);
                                strcpy(par_path,argv[i+1]);
                                par_path[length]='\0';
                        }
                        else
                        {
                                par_path=(char *)malloc(length+2);
                                strcpy(par_path,argv[i+1]);
                                par_path[length]='/';
                                par_path[length+1]='\0';
                        }
                        i=i+2;
                }		
                else
                {
                        printf("Error: don't have the parameter: %s\n",argv[i]);
                        usage();
                        exit(1);
                }
        }
//check paramters
        if(flag[0]==0)
        {
                printf("Error! Users must input the reference sequence file with -in!\n");
                usage();
                exit(1);
        }
        if(flag[1]==0)
        {
                printf("Error! Users must supply the prefix name for output file with -out!\n");
                usage();
                exit(1);
        }
        for(i=0;i<strlen(prefix);i++)
        {
                if(prefix[i]=='/')
                {
                        printf("Error! the -out parameter couldn't contain any directory!\n");
                        usage();
                        exit(1);
                }
        }
//prepare
	temp=(char *)malloc(4096);
        memset(temp,'\0',4096);
        getcwd(temp,4096);
        length=strlen(temp);
        curren_path=(char *)malloc(length+1);
        memset(curren_path,'\0',length+1);
        strcpy(curren_path,temp);
        if(flag[2]==0)
        {
                store_path=(char *)malloc(length+2);
                memset(store_path,'\0',length+2);
                strcpy(store_path,curren_path);
                store_path[length]='/';
        }
        free(temp);

//secondary structure
	if(flag[8]&&flag[9]==0)
        {
                length=strlen(curren_path);
                par_path=(char *)malloc(length+10);
                memset(par_path,'\0',length+10);
                strcpy(par_path,curren_path);
                i=length-1;
                while(par_path[i]!='/'&&i>=0)
                {
                        par_path[i]='\0';
                        i--;
                }
                strcat(par_path,"Par/");
        }
//stability parameter file
        length=strlen(par_path);
        stab_path=(char *)malloc(length+30);
        memset(stab_path,'\0',length+30);
        strcpy(stab_path,par_path);
        strcat(stab_path,"stab_parameter.txt");
//tm parameter file
        tm_path=(char *)malloc(length+30);
        memset(tm_path,'\0',length+30);
        strcpy(tm_path,par_path);
        strcat(tm_path,"tm_nn_parameter.txt");

	if(flag[8])
	{
		NumL[0]=get_num_line(par_path,0);
	        NumL[1]=get_num_line(par_path,1);
	        H_parameter=(double *)malloc((5730+2*NumL[0]+2*NumL[1])*sizeof(double));
	        memset(H_parameter,'\0',(5730+2*NumL[0]+2*NumL[1])*sizeof(double));
	        Pchar=(char *)malloc(10*NumL[0]+12*NumL[1]);
	        memset(Pchar,'\0',10*NumL[0]+12*NumL[1]);

		getStack(par_path,H_parameter);
	        getStackint2(par_path,H_parameter);
	        getDangle(par_path,H_parameter);
	        getLoop(par_path,H_parameter);
	        getTstack(par_path,H_parameter);
	        getTstack2(par_path,H_parameter);
	        getTriloop(par_path,H_parameter,Pchar,NumL);
	        getTetraloop(par_path,H_parameter,Pchar,NumL);
	        tableStartATS(6.9,H_parameter);
	        tableStartATH(2200.0,H_parameter);

		hipMemcpyToSymbol(HIP_SYMBOL(d_NumL),NumL,2*sizeof(int));
		hipMemcpyToSymbol(HIP_SYMBOL(d_Pchar),Pchar,10*NumL[0]+12*NumL[1]);
		hipMemcpyToSymbol(HIP_SYMBOL(parameter),H_parameter,(5730+2*NumL[0]+2*NumL[1])*sizeof(double));

		hipMalloc((void **)&d_numSeq,54*thread*block*sizeof(char));
		hipMalloc((void **)&d_DPT,1331*thread*block*sizeof(double));
		hipMalloc((void **)&d_ps,119*thread*block*sizeof(int));
	}

//input reference sequence
        if(access(input,0)==-1)
        {
                printf("Error! Don't have the %s file.\n",input);
                exit(1);
        }
        stat(input,&statbuf);
        length=statbuf.st_size;
        length=length+100;
        temp=(char *)malloc(length);
        memset(temp,'\0',length);
        seq=(char *)malloc(length*sizeof(char));
        memset(seq,'\0',length*sizeof(char));

        fp=fopen(input,"r");   //open the sequence file
        if(fp==NULL)
        {
                printf("Error! can't open the %s file!\n",input);
                exit(1);
        }
        fread(temp,length*sizeof(char),1,fp);
        fclose(fp); 

        length=0;
        i=0;
        while(temp[i]!='\n')
        {
                i++;
        }
        i++;
        while(temp[i]!='\0')
        {
                if(temp[i]=='\n')
                {
                        i++;
                        continue;
                }
		if(temp[i]=='a'||temp[i]=='A')
                        seq[length]='A';
                else if(temp[i]=='t'||temp[i]=='T')
                        seq[length]='T';
                else if(temp[i]=='c'||temp[i]=='C')
                        seq[length]='C';
                else if(temp[i]=='g'||temp[i]=='G')
                        seq[length]='G';
                else
                        seq[length]='N';
                i++;
                length++;
        }
        free(temp);
        length=strlen(seq);

//input Tm parameter
        fp=fopen(tm_path,"r");  //read the paramter of deltah and deltas
        if(fp==NULL)
        {
                printf("Error: can't open the %s file!\n",tm_path);
                exit(1);
        }
        while(fscanf(fp,"%d\t%f\t%f",&i,&temp1,&temp2)!=EOF)
        {
                deltah[i]=temp1;
                deltas[i]=temp2;
        }
        fclose(fp);

//input stability parameter
        fp=fopen(stab_path,"r");  //read the parameters of stability
        if(fp==NULL)
        {
                printf("Error: can't open the %s file!\n",stab_path);
                exit(1);
        }
        while(fscanf(fp,"%d\t%f",&i,&temp1)!=EOF)
        {
                stab[i]=temp1;
        }
        fclose(fp);

	Num[0]=0;
	Num[1]=0;
	Num[2]=0;
	hipMalloc((void **)&d_seq,length*sizeof(char));
	hipMemset(d_seq,'\0',length*sizeof(char));

	/////from cpu to gpu
	hipMemcpy(d_seq,seq,length*sizeof(char),hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_deltah),deltah,16*sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(d_deltas),deltas,16*sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(d_stab),stab,4096*sizeof(float));

	hipMalloc((void **)&d_len,11*length*sizeof(int));
	hipMemset(d_len,'\0',11*length*sizeof(int));
	hipMalloc((void **)&d_rev_len,11*length*sizeof(int));
        hipMemset(d_rev_len,'\0',11*length*sizeof(int));
	hipMalloc((void **)&d_Tm,11*length*sizeof(float));
	hipMemset(d_Tm,'\0',11*length*sizeof(float));

	len=(int *)malloc(11*length*sizeof(int));
	memset(len,'\0',11*length*sizeof(int));
        rev_len=(int *)malloc(11*length*sizeof(int));
        memset(rev_len,'\0',11*length*sizeof(int));
	h_Tm=(float *)malloc(11*length*sizeof(float));
	memset(h_Tm,'\0',11*length*sizeof(float));

	end=time(NULL);
	printf("It takes %d seconds to prepare.\n",(int)difftime(end,start));
	start=time(NULL);
	candidate_primer<<<block,thread>>>(d_seq,d_len,d_rev_len,flag[7],length,flag[8],d_numSeq,d_DPT,d_ps,d_Tm);
       	hipMemcpy(len,d_len,11*length*sizeof(int),hipMemcpyDeviceToHost);
        hipMemcpy(rev_len,d_rev_len,11*length*sizeof(int),hipMemcpyDeviceToHost);
	hipMemcpy(h_Tm,d_Tm,11*length*sizeof(float),hipMemcpyDeviceToHost);

	create_file(prefix,store_path,seq,len,rev_len,length,flag[7],Num,h_Tm);

	hipFree(d_len);
	hipFree(d_rev_len);
	hipFree(d_seq);
	hipFree(d_Tm);
        free(len);
        free(rev_len);
	free(seq);
	free(h_Tm);

	printf("There ara %d candidate primers used as F3/F2/B2/B3.\n",Num[1]);
        printf("There are %d candidate primers used as F1c/B1c.\n",Num[0]);
        if(flag[7]==1)
                printf("There are %d candidate primers used as LF/LB.\n",Num[2]);
        //check
        if(Num[1]<4)
                printf("Warning: there don't have enough primers(>=4) used as F3/F2/B2/B3.\n");
        if(Num[0]<2)
                printf("Warning: there don't have enough primers(>=2) used as F1c/B1c.\n");
        if(flag[7]==1 && Num[2]<1)
                printf("Warning: there don't have enough primers(>=1) used as LF/LB. But you can design LAMP primers without loop primer.\n");
	end=time(NULL);
        printf("It takes %d seconds to identify candidate single primer regions.\n",(int)difftime(end,start));

	free(store_path);
	free(prefix);
	free(stab_path);
	free(tm_path);
	free(curren_path);
	free(input);
	if(flag[8])
	{
		free(Pchar);
		free(H_parameter);
		hipFree(d_numSeq);
		hipFree(d_DPT);
		hipFree(d_ps);
	}
	if(flag[8]||flag[9])
		free(par_path);
	return 1;
}
